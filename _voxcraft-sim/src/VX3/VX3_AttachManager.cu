#include "hip/hip_runtime.h"
#include "VX3_AttachManager.h"

#include "VX3_Voxel.h"
#include "VX3_VoxelyzeKernel.cuh"

__device__ VX3_AttachManager::VX3_AttachManager(VX3_VoxelyzeKernel *k) { d_kernel = k; }

__device__ bool VX3_AttachManager::attachWhileCollide(VX3_Voxel *voxel1, VX3_Voxel *voxel2) {
    // Simple preliminary filter
    //// determined by formula
    if (!voxel1->enableAttach || !voxel2->enableAttach)
        return false;
    //// fixed voxels, no need to look further for attachment
    if (voxel1->mat->fixed || voxel2->mat->fixed)
        return false;
    //// different material, no need to attach
    if (voxel1->mat != voxel2->mat)
        return false;
    //// sticky or not
    if (!voxel1->mat->sticky)
        return false;

    // sam:
    if (voxel1->nonStickTimer > d_kernel->currentTime || voxel2->nonStickTimer > d_kernel->currentTime)
        return false;

    // Check VoxelGroup map for compatible
    int linkdir_1, linkdir_2;

    if (voxel1->d_group == voxel2->d_group) {
        // Same group, but please attach neighbors in the same group.
        VX3_Vec3D<int> diff = voxel1->groupPosition - voxel2->groupPosition;
        if (diff.Length2() > 1)
            return false;
        linkdir_1 = (diff.x == 1 ? 1 : 0) + (diff.y == 1 ? 3 : 0) + (diff.y == -1 ? 2 : 0) + (diff.z == 1 ? 5 : 0) + (diff.z == -1 ? 4 : 0);
        linkdir_2 = oppositeDirection(linkdir_1);

    } else {
        // Different groups, check for compatibility
        if (!voxel1->d_group->isCompatible(voxel1, voxel2, &linkdir_1, &linkdir_2))
            return false;
    }

    // try only form one link
    if (OnlyFormOneLink) {
        if (totalLinksFormed >= 1)
            return false;
    }
    return tryToAttach(voxel1, linkdir_1, voxel2, linkdir_2);
}


__device__ bool VX3_AttachManager::attachForNewVoxel(VX3_Voxel *voxel1, int linkdir_1, VX3_Voxel *voxel2, int linkdir_2) {
    while(1) {
        if (tryToAttach(voxel1, linkdir_1, voxel2, linkdir_2)) {
            break;
        }
    }
    return true;
}

__device__ bool VX3_AttachManager::tryToAttach(VX3_Voxel *voxel1, int linkdir_1, VX3_Voxel *voxel2, int linkdir_2) {

    bool ret = false;
    // Start Attach!
    // Only once attchment at a time, other potential attachments should be ignored, no wait.
    if (atomicCAS(&attachmentMutex, 0, 1) == 0) {
        // Entering Critical Area
        if (!voxel1->d_group->needUpdate && !voxel2->d_group->needUpdate) { // to avoid two voxels attach at the same position
            if (voxel1->links[linkdir_1] == NULL && voxel2->links[linkdir_2] == NULL) {
                VX3_Link *pL;
                pL = new VX3_Link(voxel1, (linkDirection)linkdir_1, voxel2, (linkDirection)linkdir_2, d_kernel); // make the new link (change to both materials, etc.
                if (!pL) {
                    printf(COLORCODE_BOLD_RED "ERROR: Out of memory. Link not created.\n");
                } else {
                    // update voxel1's group, it will set all connected voxels' group to voxel1's
                    voxel1->d_group->hasNewLink++;
                    voxel1->d_group->needUpdate = 1;
                    d_kernel->d_voxel_to_update_group.push_back(voxel1);

                    // if (voxel1->d_group != voxel2->d_group) {
                    //     voxel2->d_group->switchAllVoxelsTo(voxel1->d_group);
                    //     // voxel1->d_group->updateGroup(voxel1);
                    // }
                    pL->isNewLink = d_kernel->SafetyGuard;
                    d_kernel->d_v_links.push_back(pL); // add to the list

                    d_kernel->isSurfaceChanged = true;
                    // DEBUG_PRINT("%f) New Link formed.\n", d_kernel->currentTime);
                    // d_kernel->EnableCilia = false; // for debug: no cilia after attachment.
                    ret = true;
                    totalLinksFormed++;
                    
                    // sam:
                    voxel1->targetPos.clear();
                    voxel2->targetPos.clear();

                }
            }
        }
        atomicExch(&attachmentMutex, 0);
    }

    return ret;
}