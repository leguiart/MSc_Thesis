#include "hip/hip_runtime.h"
#include "VX3_MaterialLink.h"
#include "VX3_VoxelyzeKernel.cuh"

VX3_MaterialLink::VX3_MaterialLink(CVX_MaterialLink* p, VX3_VoxelyzeKernel* k):
VX3_MaterialVoxel((CVX_MaterialVoxel *)p, k),
_a1(p->_a1), _a2(p->_a2), _b1(p->_b1), _b2(p->_b2), _b3(p->_b3),
_sqA1(p->_sqA1), _sqA2xIp(p->_sqA2xIp), _sqB1(p->_sqB1), 
_sqB2xFMp(p->_sqB2xFMp), _sqB3xIp(p->_sqB3xIp) {
	for (int i=0;i<k->h_voxelMats.size();i++) {
		if (k->h_voxelMats[i] == p->vox1Mat) {
			vox1Mat = &k->d_voxelMats[i];
		} //vox1Mat and vox2Mat could be the same Mat
		if (k->h_voxelMats[i] == p->vox2Mat) {
			vox2Mat = &k->d_voxelMats[i];
		}
	}
	if (vox1Mat==NULL || vox2Mat==NULL) {
		printf(COLORCODE_BOLD_RED "ERROR: Cannot find voxel for voxelMats.\n");
	}
}
VX3_MaterialLink::~VX3_MaterialLink() {
}

__device__ VX3_MaterialLink::VX3_MaterialLink(VX3_MaterialVoxel* mat1, VX3_MaterialVoxel* mat2)
{
	vox1Mat = mat1;
	vox2Mat = mat2;

	updateAll();

}

__device__ VX3_MaterialLink& VX3_MaterialLink::operator=(const VX3_MaterialLink& vIn)
{
	VX3_MaterialVoxel::operator=(vIn); //set base VX3_MaterialVoxel class variables equal

	vox1Mat = vIn.vox1Mat;
	vox2Mat = vIn.vox2Mat;
	_a1 = vIn._a1;
	_a2 = vIn._a2;
	_b1 = vIn._b1;
	_b2 = vIn._b2;
	_b3 = vIn._b3;
	_sqA1 = vIn._sqA1;
	_sqA2xIp = vIn._sqA2xIp;
	_sqB1 = vIn._sqB1;
	_sqB2xFMp = vIn._sqB2xFMp;
	_sqB3xIp = vIn._sqB3xIp;

	return *this;
}

__device__ bool VX3_MaterialLink::updateAll()
{
	nomSize = 0.5*(vox1Mat->nomSize + vox2Mat->nomSize); //these should be the same...

	r=(int)(0.5*(vox1Mat->r + vox2Mat->r));
	g=(int)(0.5*(vox1Mat->g + vox2Mat->g));
	b=(int)(0.5*(vox1Mat->b + vox2Mat->b));
	a=(int)(0.5*(vox1Mat->a + vox2Mat->a));

	rho = 0.5f*(vox1Mat->rho + vox2Mat->rho);
	alphaCTE = 0.5f*(vox1Mat->alphaCTE + vox2Mat->alphaCTE);
	muStatic = 0.5f*(vox1Mat->muStatic + vox2Mat->muStatic);
	muKinetic = 0.5f*(vox1Mat->muKinetic + vox2Mat->muKinetic);
	zetaInternal = 0.5f*(vox1Mat->zetaInternal + vox2Mat->zetaInternal);
	zetaGlobal = 0.5f*(vox1Mat->zetaGlobal + vox2Mat->zetaGlobal);
	zetaCollision= 0.5f*(vox1Mat->zetaCollision + vox2Mat->zetaCollision);

	extScale=VX3_Vec3D<>(1.0, 1.0, 1.0);

	//failure stress (f) is the minimum of the two failure stresses, or if both are -1.0f it should also be -1.0f to denote no failure specified
	float stressFail=-1.0f, /*strainFail=-1.0f,*/ f1=vox1Mat->sigmaFail, f2=vox2Mat->sigmaFail;
	if (f1 == -1.0f) stressFail = f2; //-1.0f or vox2Mat fail
	else if (f2 == -1.0f) stressFail = f1; //vox1Mat fail
	else stressFail = f1 < f2 ? f1 : f2; //the lesser stress denotes failure

	if (vox1Mat->linear && vox2Mat->linear) setModelLinear(2.0f*vox1Mat->E*vox2Mat->E/(vox1Mat->E+vox2Mat->E), stressFail);
	else { //at least 1 bilinear or data-based, so build up data points and apply it.
		VX3_dVector<float> newStressValues, newStrainValues;
		newStressValues.push_back(0.0f);
		newStrainValues.push_back(0.0f);

		//step up through ascending strains data points (could alternate randomly between vox1Mat and vox2Mat points
		int dataIt1 = 1, dataIt2 = 1; //iterators through each data point of the model
		while (dataIt1 < (int)vox1Mat->d_strainData.size() && dataIt2 < (int)vox2Mat->d_strainData.size()){
			float strain = FLT_MAX; //strain for the next data point is the smaller of the two possible next strain points (but we have to make sure we don't access off the end of one of the arrays)
			if (dataIt1 < (int)vox1Mat->d_strainData.size()) strain = vox1Mat->d_strainData[dataIt1];
			if (dataIt2 < (int)vox2Mat->d_strainData.size() && vox2Mat->d_strainData[dataIt2]<strain) strain = vox2Mat->d_strainData[dataIt2];
			else assert(strain != FLT_MAX); //this should never happen

			if (strain == vox1Mat->d_strainData[dataIt1]) dataIt1++;
			if (strain == vox2Mat->d_strainData[dataIt2]) dataIt2++;


			float modulus1 = vox1Mat->modulus(strain-FLT_EPSILON);
			float modulus2 = vox2Mat->modulus(strain-FLT_EPSILON);
			float thisModulus = 2.0f*modulus1*modulus2/(modulus1+modulus2);

			//add to the new strain/stress values
			int lastDataIndex = newStrainValues.size()-1;

			newStrainValues.push_back(strain);
			newStressValues.push_back(newStressValues[lastDataIndex] + thisModulus*(strain - newStrainValues[lastDataIndex])); //springs in series equation
		}

		setModel(newStrainValues.size(), &newStrainValues[0], &newStressValues[0]);

		//override failure points in case no failure was specified before (as possible in combos of linear and bilinear materials)
		//yield point is handled correctly in setModel.
		sigmaFail = stressFail;
		epsilonFail = stressFail==-1.0f ? -1.0f : strain(stressFail);
	}

	//poissons ratio: choose such that Ehat ends up according to spring in series of Ehat1 and EHat2
	if (vox1Mat->nu==0 && vox2Mat->nu==0) nu = 0;
	else { //poissons ratio: choose such that Ehat ends up according to spring in series of Ehat1 and EHat2
		float tmpEHat = 2*vox1Mat->_eHat*vox2Mat->_eHat/(vox1Mat->_eHat+vox2Mat->_eHat);
		float tmpE = youngsModulus();
		//completing the square algorithm to solve for nu.
		//eHat = E/((1-2nu)(1+nu)) -> E/EHat = -2nu^2-nu+1 -> nu^2+0.5nu = (EHat+E)/(2EHat)
		float c2 = (tmpEHat-tmpE)/(2*tmpEHat)+0.0625; //nu^2+0.5nu+0.0625 = c2 -> (nu+0.25)^2 = c2
		nu = sqrt(c2)-0.25; //from solving above
	}

	return updateDerived();
}

__device__ bool VX3_MaterialLink::updateDerived() 
{
	VX3_MaterialVoxel::updateDerived(); //update base VX3_Material class derived variables

	//stiffnesses terms for links
	float L = (float)nomSize;
	_a1 = E*L; //EA/L : Units of N/m
	_a2 = E * L*L*L / (12.0f*(1+nu)); //GJ/L : Units of N-m
	_b1 = E*L; //12EI/L^3 : Units of N/m
	_b2 = E*L*L/2.0f; //6EI/L^2 : Units of N (or N-m/m: torque related to linear distance)
	_b3 = E*L*L*L/6.0f; //2EI/L : Units of N-m
	
	//damping sqrt(mk) terms (with sqrt(m) factored out)
	_sqA1=sqrt(_a1);
	_sqA2xIp=sqrt(_a2*L*L/6.0f);
	_sqB1=sqrt(_b1);
	_sqB2xFMp=sqrt(_b2*L/2.0f);
	_sqB3xIp=sqrt(_b3*L*L/6.0f);

	return true;
}
