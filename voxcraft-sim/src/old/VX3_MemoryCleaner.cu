#include "VX3_MemoryCleaner.h"
// #include "VX3_TaskManager.h"

bool VX3_MemoryCleaner_running=true;
boost::mutex MemoryCleaner_mutex;
std::vector<void *> MemoryCleaner_toBeFreedCUDAPointer;

void VX3_MemoryCleaner::operator()() {
    printf("Start. %d\n", VX3_MemoryCleaner_running);
    // while(VX3_MemoryCleaner_running) {
    //     // printf("Thread count: %ld\n", TaskManager_all_threads.size());
    //     if (TaskManager_all_threads.size()>0) {
    //          // try_join_for
    //         for (int i=0;i<TaskManager_all_threads.size();i++) {
    //             if (TaskManager_all_threads[i].try_join_for( boost::chrono::nanoseconds(1) )) {
    //                 // printf("clean one thread.\n");
    //                 TaskManager_all_threads_mutex.lock();
    //                 TaskManager_all_threads.erase(TaskManager_all_threads.begin()+i);
    //                 TaskManager_all_threads_mutex.unlock();
    //             }
    //         }
    //     }
    //     if (TaskManager_all_threads.size()==0) {
    //         if (MemoryCleaner_toBeFreedCUDAPointer.size()>0) {
    //             MemoryCleaner_mutex.lock();
    //             MemoryCleaner_toBeFreedCUDAPointer.clear();
    //             MemoryCleaner_mutex.unlock();
    //             printf("hipDeviceReset() called! should release all previously used memory.\n");
    //             CUDA_ERROR_CHECK(hipDeviceReset());
    //         }
    //         // while(MemoryCleaner_toBeFreedCUDAPointer.size()>0) {
    //         //     void* pointer;
    //         //     MemoryCleaner_mutex.lock();
    //         //     pointer = MemoryCleaner_toBeFreedCUDAPointer.back();
    //         //     MemoryCleaner_toBeFreedCUDAPointer.pop_back();
    //         //     MemoryCleaner_mutex.unlock();
    //         //     // printf("Cleaning %p (%ld remain)...\n", pointer, MemoryCleaner_toBeFreedCUDAPointer.size());
    //         //     VcudaFree(pointer);
    //         //     std::this_thread::sleep_for(std::chrono::milliseconds(1));
    //         // }
    //     }
    //     std::this_thread::sleep_for(std::chrono::seconds(1));
    // }

}