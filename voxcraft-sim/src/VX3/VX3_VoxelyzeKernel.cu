#include "hip/hip_runtime.h"
#include "VX3_MemoryCleaner.h"
#include "VX3_VoxelyzeKernel.cuh"

/* Tools */
__device__ int bound(int x, int min, int max) {
    if (x < min)
        return min;
    if (x > max)
        return max;
    return x;
}

/* Sub GPU Threads */
__global__ void gpu_update_links(VX3_Link **links, int num);
__global__ void gpu_update_voxels(VX3_Voxel *voxels, int num, double dt, double currentTime, VX3_VoxelyzeKernel *k);
__global__ void gpu_update_temperature(VX3_Voxel *voxels, int num, double TempAmplitude, double TempPeriod, double currentTime, VX3_VoxelyzeKernel* k);
__global__ void gpu_update_attach(VX3_Voxel **surface_voxels, int num, double watchDistance, VX3_VoxelyzeKernel *k);
__global__ void gpu_update_cilia_force(VX3_Voxel **surface_voxels, int num, VX3_VoxelyzeKernel *k);
__global__ void gpu_clear_lookupgrid(VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid, int num);
__global__ void gpu_insert_lookupgrid(VX3_Voxel **d_surface_voxels, int num, VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid,
                                      VX3_Vec3D<> *gridLowerBound, VX3_Vec3D<> *gridDelta, int lookupGrid_n);
__global__ void gpu_collision_attachment_lookupgrid(VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid, int num, double watchDistance,
                                                    VX3_VoxelyzeKernel *k);
__global__ void gpu_update_detach(VX3_Link **links, int num, VX3_VoxelyzeKernel *k);
/* Host methods */

VX3_VoxelyzeKernel::VX3_VoxelyzeKernel(CVX_Sim *In) {

    voxSize = In->Vx.voxSize;

    num_d_voxelMats = In->Vx.voxelMats.size();
    VcudaMalloc((void **)&d_voxelMats, num_d_voxelMats * sizeof(VX3_MaterialVoxel));
    {
        // push all h first, since there will be reference below
        for (auto mat : In->Vx.voxelMats) {
            h_voxelMats.push_back(mat);
        }
        int i = 0;
        for (auto mat : In->Vx.voxelMats) {
            VX3_MaterialVoxel tmp_voxelMat(mat, this);
            VcudaMemcpy(d_voxelMats + i, &tmp_voxelMat, sizeof(VX3_MaterialVoxel), VcudaMemcpyHostToDevice);
            i++;
        }
    }

    num_d_linkMats = In->Vx.linkMats.size();
    VcudaMalloc((void **)&d_linkMats, num_d_linkMats * sizeof(VX3_MaterialLink));
    {
        int i = 0;
        std::vector<VX3_MaterialLink *> tmp_v_linkMats;
        for (CVX_MaterialLink *mat : In->Vx.linkMats) {
            // printf("mat->vox1Mat %p, mat->vox2Mat %p.\n", mat->vox1Mat,
            // mat->vox2Mat);
            VX3_MaterialLink tmp_linkMat(mat, this);
            VcudaMemcpy(d_linkMats + i, &tmp_linkMat, sizeof(VX3_MaterialLink), VcudaMemcpyHostToDevice);
            tmp_v_linkMats.push_back(d_linkMats + i);
            h_linkMats.push_back(mat);
            i++;
        }
        hd_v_linkMats = VX3_hdVector<VX3_MaterialLink *>(tmp_v_linkMats);
    }

    num_d_voxels = In->Vx.voxelsList.size();
    VcudaMalloc((void **)&d_voxels, num_d_voxels * sizeof(VX3_Voxel));
    for (int i = 0; i < num_d_voxels; i++) {
        h_voxels.push_back(In->Vx.voxelsList[i]);
        h_lookup_voxels[In->Vx.voxelsList[i]] = d_voxels + i;
    }
    VcudaMalloc((void **)&d_initialPosition, num_d_voxels * sizeof(Vec3D<>));

    num_d_links = In->Vx.linksList.size();
    std::vector<VX3_Link *> tmp_v_links;
    VcudaMalloc((void **)&d_links, num_d_links * sizeof(VX3_Link));
    VX3_Link *tmp_link_cache = (VX3_Link *)malloc(num_d_links * sizeof(VX3_Link));
    for (int i = 0; i < num_d_links; i++) {
        VX3_Link tmp_link(In->Vx.linksList[i], this);
        memcpy(tmp_link_cache + i, &tmp_link, sizeof(VX3_Link));
        tmp_v_links.push_back(d_links + i); // not copied yet, but still ok to get the address
        h_links.push_back(In->Vx.linksList[i]);
    }
    VcudaMemcpy(d_links, tmp_link_cache, num_d_links * sizeof(VX3_Link), VcudaMemcpyHostToDevice);
    hd_v_links = VX3_hdVector<VX3_Link *>(tmp_v_links);
    for (int i = 0; i < num_d_links; i++) {
        h_lookup_links[In->Vx.linksList[i]] = d_links + i;
    }

    for (int i = 0; i < num_d_voxels; i++) {
        // set values for GPU memory space
        VX3_Voxel tmp_voxel(In->Vx.voxelsList[i], this);
        VcudaMemcpy(d_voxels + i, &tmp_voxel, sizeof(VX3_Voxel), VcudaMemcpyHostToDevice);
    }

    // Not all data is in Vx, here are others:
    DtFrac = In->DtFrac;
    StopConditionType = In->StopConditionType;
    StopConditionValue = In->StopConditionValue;
    TempEnabled = In->pEnv->TempEnabled;
    VaryTempEnabled = In->pEnv->VaryTempEnabled;
    TempBase = In->pEnv->TempBase;
    TempAmplitude = In->pEnv->TempAmplitude;
    TempPeriod = In->pEnv->TempPeriod;
    // currentTemperature = TempBase + TempAmplitude;

    d_surface_voxels = NULL;
}

void VX3_VoxelyzeKernel::cleanup() {
    // The reason not use ~VX3_VoxelyzeKernel is that will be automatically call
    // multiple times after we use memcpy to clone objects.
    MycudaFree(d_linkMats);
    MycudaFree(d_voxels);
    MycudaFree(d_links);
    // MycudaFree(d_collisionsStale);
    if (d_surface_voxels) {
        MycudaFree(d_surface_voxels); // can __device__ malloc pointer be freed
                                      // by hipFree in __host__??
    }
    // MycudaFree(d_collisions);
}

/* Cuda methods : cannot use any CVX_xxx, and no std::, no boost::, and no
 * filesystem. */

__device__ void VX3_VoxelyzeKernel::syncVectors() {
    d_v_linkMats.clear();
    d_v_collisions.clear();
    d_targets.clear();
    // allocate memory for collision lookup table
    num_lookupGrids = lookupGrid_n * lookupGrid_n * lookupGrid_n;
    d_collisionLookupGrid = (VX3_dVector<VX3_Voxel *> *)malloc(num_lookupGrids * sizeof(VX3_dVector<VX3_Voxel *>));
    if (d_collisionLookupGrid == NULL) {
        printf(COLORCODE_BOLD_RED "ERROR: not enough memory.\n");
    }
    for (int i = 0; i < hd_v_linkMats.size(); i++) {
        d_v_linkMats.push_back(hd_v_linkMats[i]);
    }

    d_v_links.clear();
    for (int i = 0; i < hd_v_links.size(); i++) {
        d_v_links.push_back(hd_v_links[i]);
    }

    for (int i = 0; i < num_d_voxelMats; i++) {
        d_voxelMats[i].syncVectors();
    }

    for (int i = 0; i < num_d_linkMats; i++) {
        d_linkMats[i].syncVectors();
    }

    for (int i = 0; i < num_d_voxels; i++) {
        d_voxels[i].syncVectors();
    }
}
__device__ void VX3_VoxelyzeKernel::saveInitialPosition() {
    for (int i = 0; i < num_d_voxels; i++) {
        d_initialPosition[i] = d_voxels[i].pos;
        // Save this value to voxel, so it can be read out when collecting results in cpu.
        d_voxels[i].isMeasured = (bool) d_voxels[i].mat->isMeasured;
    }
}
__device__ bool VX3_VoxelyzeKernel::StopConditionMet(void) // have we met the stop condition yet?
{
    if (VX3_MathTree::eval(currentCenterOfMass.x, currentCenterOfMass.y, currentCenterOfMass.z, collisionCount, currentTime, recentAngle,
                           targetCloseness, numClosePairs, num_d_voxels, StopConditionFormula) > 0) {
        // double a =
        //     VX3_MathTree::eval(currentCenterOfMass.x, currentCenterOfMass.y, currentCenterOfMass.z, collisionCount, currentTime,
        //     StopConditionFormula);
        // printf("stop score: %f.\n\n", a);
        return true;
    }
    if (forceExit)
        return true;
    return false;
    // if (StopConditionType != SC_MAX_SIM_TIME) {
    //     printf(COLORCODE_BOLD_RED "StopConditionType: %d. Type of stop condition no supported for "
    //                               "now.\n" COLORCODE_RESET,
    //            StopConditionType);
    //     return true;
    // }
    // return currentTime > StopConditionValue ? true : false;
}

__device__ double VX3_VoxelyzeKernel::recommendedTimeStep() {
    // find the largest natural frequency (sqrt(k/m)) that anything in the
    // simulation will experience, then multiply by 2*pi and invert to get the
    // optimally largest timestep that should retain stability
    double MaxFreq2 = 0.0f; // maximum frequency in the simulation in rad/sec
    if (!num_d_links) {
        printf("WARNING: No links.\n");
    }
    if (!num_d_voxels) {
        printf(COLORCODE_BOLD_RED "ERROR: No voxels.\n");
    }
    for (int i = 0; i < num_d_links; i++) {
        VX3_Link *pL = d_links + i;
        // axial
        double m1 = pL->pVNeg->mat->mass(), m2 = pL->pVPos->mat->mass();
        double thisMaxFreq2 = pL->axialStiffness() / (m1 < m2 ? m1 : m2);
        if (thisMaxFreq2 > MaxFreq2)
            MaxFreq2 = thisMaxFreq2;
        // rotational will always be less than or equal
    }
    if (MaxFreq2 <= 0.0f) {                      // didn't find anything (i.e no links) check for
                                                 // individual voxelss
        for (int i = 0; i < num_d_voxels; i++) { // for each link
            double thisMaxFreq2 = d_voxels[i].mat->youngsModulus() * d_voxels[i].mat->nomSize / d_voxels[i].mat->mass();
            if (thisMaxFreq2 > MaxFreq2)
                MaxFreq2 = thisMaxFreq2;
        }
    }
    if (MaxFreq2 <= 0.0f)
        return 0.0f;
    else
        return 1.0f / (6.283185f * sqrt(MaxFreq2)); // the optimal timestep is to advance one
                                                    // radian of the highest natural frequency
}

__device__ void VX3_VoxelyzeKernel::updateTemperature() {
    // updates the temperatures For Actuation!
    // different temperatures in different objs are not support for now.
    if (VaryTempEnabled) {
        if (TempPeriod > 0) {
            int blockSize = 512;
            int minGridSize;
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_temperature, 0,
                                               num_d_voxels); // Dynamically calculate blockSize
            int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize;
            int blockSize_voxels = num_d_voxels < blockSize ? num_d_voxels : blockSize;
            gpu_update_temperature<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, num_d_voxels, TempAmplitude, TempPeriod, currentTime, this);
            CUDA_CHECK_AFTER_CALL();
            VcudaDeviceSynchronize();
        }
    }
}

__device__ bool VX3_VoxelyzeKernel::doTimeStep(float dt) {
    // clock_t time_measures[10];
    // time_measures[0] = clock();
    updateTemperature();
    CurStepCount++;
    if (dt == 0)
        return true;
    else if (dt < 0) {
        if (!OptimalDt) {
            OptimalDt = recommendedTimeStep();
        }
        if (OptimalDt < 1e-10) {
            CUDA_DEBUG_LINE("recommendedTimeStep is zero.");
            OptimalDt = 1e-10;
            // return false;
        }
        dt = DtFrac * OptimalDt;
    }
    bool Diverged = false;

    int blockSize;
    int minGridSize;
    if (d_v_links.size()) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_links, 0,
                                           d_v_links.size()); // Dynamically calculate blockSize
        int gridSize_links = (d_v_links.size() + blockSize - 1) / blockSize;
        int blockSize_links = d_v_links.size() < blockSize ? d_v_links.size() : blockSize;
        // if (CurStepCount % 1000 == 0 || currentTime>1.0) {
        //     printf("&d_v_links[0] %p; d_v_links.size() %d. \n", &d_v_links[0], d_v_links.size());
        // }
        gpu_update_links<<<gridSize_links, blockSize_links>>>(&d_v_links[0], d_v_links.size());
        CUDA_CHECK_AFTER_CALL();
        VcudaDeviceSynchronize();

        // checking every link for diverge is too wasteful! using random
        // sampling.
        int r = random(d_v_links.size(), clock());
        if (d_v_links[r]->axialStrain() > 100) {
            CUDA_DEBUG_LINE("Diverged.");
            Diverged = true; // catch divergent condition! (if any thread sets
                             // true we will fail, so don't need mutex...
        }
        if (Diverged)
            return false;
    }

    if (isSurfaceChanged) {
        isSurfaceChanged = false;

        regenerateSurfaceVoxels();
    }

    if (enableAttach || EnableCollision) { // either attachment and collision need measurement for pairwise distances
        updateAttach();
    }
    if (enableDetach) {
        updateDetach();
    }

    if (EnableCilia) {
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_cilia_force, 0,
                                           num_d_surface_voxels); // Dynamically calculate blockSize
        int gridSize_voxels = (num_d_surface_voxels + blockSize - 1) / blockSize;
        int blockSize_voxels = num_d_surface_voxels < blockSize ? num_d_surface_voxels : blockSize;
        gpu_update_cilia_force<<<gridSize_voxels, blockSize_voxels>>>(d_surface_voxels, num_d_surface_voxels, this);
        CUDA_CHECK_AFTER_CALL();
        VcudaDeviceSynchronize();
    }

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_voxels, 0,
                                       num_d_voxels); // Dynamically calculate blockSize
    int gridSize_voxels = (num_d_voxels + blockSize - 1) / blockSize;
    int blockSize_voxels = num_d_voxels < blockSize ? num_d_voxels : blockSize;
    gpu_update_voxels<<<gridSize_voxels, blockSize_voxels>>>(d_voxels, num_d_voxels, dt, currentTime, this);
    CUDA_CHECK_AFTER_CALL();
    VcudaDeviceSynchronize();

    int CycleStep =
        int(TempPeriod / dt); // Sample at the same time point in the cycle, to avoid the impact of actuation as much as possible.
    if (CurStepCount % CycleStep == 0) {
        angleSampleTimes++;

        currentCenterOfMass_history[0] = currentCenterOfMass_history[1];
        currentCenterOfMass_history[1] = currentCenterOfMass;
        updateCurrentCenterOfMass();
        auto A = currentCenterOfMass_history[0];
        auto B = currentCenterOfMass_history[1];
        auto C = currentCenterOfMass;
        if (B == C || A == B || angleSampleTimes < 3) {
            recentAngle = 0; // avoid divide by zero, and don't include first two steps where A and B are still 0.
        } else {
            recentAngle = acos((B - A).Dot(C - B) / (B.Dist(A) * C.Dist(B)));
        }
        // printf("(%d) recentAngle = %f\n", angleSampleTimes, recentAngle);

        // Also calculate targetCloseness here.
        computeTargetCloseness();
    }

    if (SecondaryExperiment) {
        // SecondaryExperiment handle tags:
        // RemoveFromSimulationAfterThisManySeconds
        // ReinitializeInitialPositionAfterThisManySeconds
        // TurnOnThermalExpansionAfterThisManySeconds
        // TurnOnCiliaAfterThisManySeconds

        removeVoxels();
        if (InitialPositionReinitialized == false && ReinitializeInitialPositionAfterThisManySeconds < currentTime) {
            InitialPositionReinitialized = true;
            InitializeCenterOfMass();
            saveInitialPosition();
        }

    }

    currentTime += dt;
    // time_measures[1] = clock();
    // printf("running time for each step: \n");
    // for (int i=0;i<1;i++)
    //     printf("\t%d) %ld clock cycles.\n", i,
    //     time_measures[i+1]-time_measures[i]);
    return true;
}

__device__ void VX3_VoxelyzeKernel::InitializeCenterOfMass() {
    initialCenterOfMass = currentCenterOfMass;
}

__device__ void VX3_VoxelyzeKernel::removeVoxels() {
    for (int i=0;i<num_d_voxelMats;i++) {
        if (d_voxelMats[i].removed == false &&
        d_voxelMats[i].RemoveFromSimulationAfterThisManySeconds > 0 &&
        d_voxelMats[i].RemoveFromSimulationAfterThisManySeconds < currentTime ) {
            VX3_Voxel* neighbor_voxel;

            for (int j=0;j<num_d_voxels;j++) {
                if (d_voxels[j].mat == &d_voxelMats[i] && d_voxels[j].removed == false) {
                    d_voxels[j].removed = true; // mark this voxel as removed
                    for (int k=0;k<6;k++) { // check links in all direction
                        if (d_voxels[j].links[k]) {
                            d_voxels[j].links[k]->removed = true; // mark the link as removed
                            if (d_voxels[j].links[k]->pVNeg == &d_voxels[j]) { // this voxel is pVNeg
                                neighbor_voxel = d_voxels[j].links[k]->pVPos;
                            } else {
                                neighbor_voxel = d_voxels[j].links[k]->pVNeg;
                            }
                            for (int m=0;m<6;m++) {
                                if (neighbor_voxel->links[m] == d_voxels[j].links[k]) {
                                    neighbor_voxel->links[m] = NULL; // delete the neighbor's link
                                    break;
                                }
                            }
                            d_voxels[j].links[k] = NULL; // delete this voxel's link
                        }
                    }
                }
            }
            d_voxelMats[i].removed = true;
            isSurfaceChanged = true;
        }
    }

}

__device__ void VX3_VoxelyzeKernel::updateAttach() {
    // for each surface voxel pair, check distance < watchDistance, make a new
    // link between these two voxels, updateSurface().
    int blockSize;
    int minGridSize;
    if (false) {
        // the parameters of grid are set in gpu_update_voxels, so detection only useful after initialization
        if (gridLowerBound != gridUpperBound) {
            gridDelta = (gridUpperBound - gridLowerBound) / lookupGrid_n;
            if (gridDelta.x < voxSize * 2) {
                gridDelta.x = voxSize * 2;
            }
            if (gridDelta.y < voxSize * 2) {
                gridDelta.y = voxSize * 2;
            }
            if (gridDelta.z < voxSize * 2) {
                gridDelta.z = voxSize * 2;
            }
            // printf("gridLowerBound (%f,%f,%f), gridDelta (%f,%f,%f), gridUpperBound (%f,%f,%f).\n\n", gridLowerBound.x, gridLowerBound.y,
            //        gridLowerBound.z, gridDelta.x, gridDelta.y, gridDelta.z, gridUpperBound.x, gridUpperBound.y, gridUpperBound.z);
            // clear all lookupGrids
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_clear_lookupgrid, 0,
                                               num_lookupGrids); // Dynamically calculate blockSize
            int gridSize_voxels = (num_lookupGrids + blockSize - 1) / blockSize;
            int blockSize_voxels = num_lookupGrids < blockSize ? num_lookupGrids : blockSize;
            gpu_clear_lookupgrid<<<gridSize_voxels, blockSize_voxels>>>(d_collisionLookupGrid, num_lookupGrids);
            CUDA_CHECK_AFTER_CALL();
            VcudaDeviceSynchronize();
            // build lookupGrids: put surface voxels into grids
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_insert_lookupgrid, 0,
                                               num_d_surface_voxels); // Dynamically calculate blockSize
            gridSize_voxels = (num_d_surface_voxels + blockSize - 1) / blockSize;
            blockSize_voxels = num_d_surface_voxels < blockSize ? num_d_surface_voxels : blockSize;
            gpu_insert_lookupgrid<<<gridSize_voxels, blockSize_voxels>>>(d_surface_voxels, num_d_surface_voxels, d_collisionLookupGrid,
                                                                         &gridLowerBound, &gridDelta, lookupGrid_n);
            CUDA_CHECK_AFTER_CALL();
            VcudaDeviceSynchronize();
            // detect collision: voxels in each grid with voxels within this grid and its neighbors
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_collision_attachment_lookupgrid, 0,
                                               num_lookupGrids); // Dynamically calculate blockSize
            gridSize_voxels = (num_lookupGrids + blockSize - 1) / blockSize;
            blockSize_voxels = num_lookupGrids < blockSize ? num_lookupGrids : blockSize;
            gpu_collision_attachment_lookupgrid<<<gridSize_voxels, blockSize_voxels>>>(d_collisionLookupGrid, num_lookupGrids,
                                                                                       watchDistance, this);
            CUDA_CHECK_AFTER_CALL();
            VcudaDeviceSynchronize();
        }
    } else {
        // Pairwise detection O(n ^ 2)
        blockSize = 16;
        dim3 dimBlock(blockSize, blockSize);
        dim3 dimGrid((num_d_surface_voxels + dimBlock.x - 1) / dimBlock.x, (num_d_surface_voxels + dimBlock.y - 1) / dimBlock.y);
        // printf("num_d_surface_voxels %d\n", num_d_surface_voxels);
        gpu_update_attach<<<dimGrid, dimBlock>>>(d_surface_voxels, num_d_surface_voxels, watchDistance,
                                                 this); // invoke two dimensional gpu threads 'CUDA C++ Programming
                                                        // Guide', Nov 2019, P52.
        CUDA_CHECK_AFTER_CALL();
    }
}

__device__ void VX3_VoxelyzeKernel::updateDetach() {
    if (d_v_links.size()) {
        int minGridSize, blockSize;
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gpu_update_detach, 0,
                                           d_v_links.size()); // Dynamically calculate blockSize
        int gridSize_links = (d_v_links.size() + blockSize - 1) / blockSize;
        int blockSize_links = d_v_links.size() < blockSize ? d_v_links.size() : blockSize;
        // if (CurStepCount % 1000 == 0 || currentTime>1.0) {
        //     printf("&d_v_links[0] %p; d_v_links.size() %d. \n", &d_v_links[0], d_v_links.size());
        // }
        gpu_update_detach<<<gridSize_links, blockSize_links>>>(&d_v_links[0], d_v_links.size(), this);
        CUDA_CHECK_AFTER_CALL();
        VcudaDeviceSynchronize();
    }
}

__device__ void VX3_VoxelyzeKernel::updateCurrentCenterOfMass() {
    double TotalMass = 0;
    VX3_Vec3D<> Sum(0, 0, 0);
    for (int i = 0; i < num_d_voxels; i++) {
        if (!d_voxels[i].mat->isMeasured) {
            continue;
        }
        double ThisMass = d_voxels[i].material()->mass();
        Sum += d_voxels[i].position() * ThisMass;
        TotalMass += ThisMass;
    }
    if (TotalMass==0) {
        currentCenterOfMass = VX3_Vec3D<>();
        return;
    }
    currentCenterOfMass = Sum / TotalMass;
}

__device__ void VX3_VoxelyzeKernel::regenerateSurfaceVoxels() {
    // regenerate d_surface_voxels
    if (d_surface_voxels) {
        delete d_surface_voxels;
        d_surface_voxels = NULL;
    }
    VX3_dVector<VX3_Voxel *> tmp;
    for (int i = 0; i < num_d_voxels; i++) {
        d_voxels[i].updateSurface();
        if (d_voxels[i].isSurface() && !d_voxels[i].removed) {
            tmp.push_back(&d_voxels[i]);
        }
    }
    num_d_surface_voxels = tmp.size();
    d_surface_voxels = (VX3_Voxel **)malloc(num_d_surface_voxels * sizeof(VX3_Voxel));
    for (int i = 0; i < num_d_surface_voxels; i++) {
        d_surface_voxels[i] = tmp[i];
    }
}

__device__ VX3_MaterialLink *VX3_VoxelyzeKernel::combinedMaterial(VX3_MaterialVoxel *mat1, VX3_MaterialVoxel *mat2) {
    for (int i = 0; i < d_v_linkMats.size(); i++) {
        VX3_MaterialLink *thisMat = d_v_linkMats[i];
        if ((thisMat->vox1Mat == mat1 && thisMat->vox2Mat == mat2) || (thisMat->vox1Mat == mat2 && thisMat->vox2Mat == mat1))
            return thisMat; // already exist
    }

    VX3_MaterialLink *newMat = new VX3_MaterialLink(mat1, mat2); // where to free this?
    d_v_linkMats.push_back(newMat);
    mat1->d_dependentMaterials.push_back(newMat);
    mat2->d_dependentMaterials.push_back(newMat);

    return newMat;
}

__device__ void VX3_VoxelyzeKernel::computeFitness() {
    VX3_Vec3D<> offset = currentCenterOfMass - initialCenterOfMass;
    fitness_score = VX3_MathTree::eval(offset.x, offset.y, offset.z, collisionCount, currentTime, recentAngle, targetCloseness,
                                       numClosePairs, num_d_voxels, fitness_function);
}

__device__ void VX3_VoxelyzeKernel::registerTargets() {
    for (int i = 0; i < num_d_voxels; i++) {
        auto v = &d_voxels[i];
        if (v->mat->isTarget) {
            d_targets.push_back(v);
        }
    }
}

__device__ void VX3_VoxelyzeKernel::computeTargetCloseness() {
    // this function is called periodically. not very often. once every thousands of steps.
    if (MaxDistInVoxelLengthsToCountAsPair==0)
        return;
    double R = MaxDistInVoxelLengthsToCountAsPair * voxSize;
    double ret = 0;
    numClosePairs = 0;
    for (int i = 0; i < d_targets.size(); i++) {
        for (int j = i + 1; j < d_targets.size(); j++) {
            double distance = d_targets[i]->pos.Dist(d_targets[j]->pos);
            if (distance < R) {
                numClosePairs++;
            }
            ret += 1 / distance;
        }
    }
    targetCloseness = ret;
    // printf("targetCloseness: %f\n", targetCloseness);
}

/* Sub GPU Threads */
__global__ void gpu_update_links(VX3_Link **links, int num) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        VX3_Link *t = links[gindex];
        if (t->removed)
            return;
        if (t->pVPos->mat->fixed && t->pVNeg->mat->fixed)
            return;
        if (t->isDetached)
            return;
        t->updateForces();
        if (t->axialStrain() > 100) {
            printf(COLORCODE_BOLD_RED "ERROR: Diverged.");
        }
    }
}
__global__ void gpu_update_voxels(VX3_Voxel *voxels, int num, double dt, double currentTime, VX3_VoxelyzeKernel *k) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        VX3_Voxel *t = &voxels[gindex];
        if (t->removed)
            return;
        if (t->mat->fixed)
            return; // fixed voxels, no need to update position
        t->timeStep(dt, currentTime, k);

        // update lower bound and upper bound
        if (t->pos.x < k->gridLowerBound.x) {
            k->gridLowerBound.x = t->pos.x;
        } else if (t->pos.x > k->gridUpperBound.x) {
            k->gridUpperBound.x = t->pos.x;
        }
        if (t->pos.y < k->gridLowerBound.y) {
            k->gridLowerBound.y = t->pos.y;
        } else if (t->pos.y > k->gridUpperBound.y) {
            k->gridUpperBound.y = t->pos.y;
        }
        if (t->pos.z < k->gridLowerBound.z) {
            k->gridLowerBound.z = t->pos.z;
        } else if (t->pos.z > k->gridUpperBound.z) {
            k->gridUpperBound.z = t->pos.z;
        }
        // update sticky status
        t->enableAttach = false;
        if (VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[0]) > 0 &&
            VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[1]) > 0 &&
            VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[2]) > 0 &&
            VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[3]) > 0 &&
            VX3_MathTree::eval(t->pos.x, t->pos.y, t->pos.z, k->collisionCount, currentTime, k->recentAngle, k->targetCloseness,
                               k->numClosePairs, k->num_d_voxels, k->AttachCondition[4]) > 0) {
            t->enableAttach = true;
        };
    }
}

__global__ void gpu_update_temperature(VX3_Voxel *voxels, int num, double TempAmplitude, double TempPeriod, double currentTime, VX3_VoxelyzeKernel* k) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        // vfloat tmp = pEnv->GetTempAmplitude() *
        // sin(2*3.1415926f*(CurTime/pEnv->GetTempPeriod() + pV->phaseOffset)) -
        // pEnv->GetTempBase();
        VX3_Voxel *t = &voxels[gindex];
        if (t->removed)
            return;
        if (t->mat->TurnOnThermalExpansionAfterThisManySeconds > currentTime)
            return;
        if (t->mat->fixed)
            return; // fixed voxels, no need to update temperature
        double currentTemperature =
            TempAmplitude * sin(2 * 3.1415926f * (currentTime / TempPeriod + t->phaseOffset)); // update the global temperature
        // TODO: if we decide not to use PhaseOffset any more, we can move this calculation outside.
        // By default we don't enable expansion. But we can enable that in VXA.
        if (!k->EnableExpansion) {
            if (currentTemperature > 0) {
                currentTemperature = 0;
            }
        }
        t->setTemperature(currentTemperature);
        // t->setTemperature(0.0f);
    }
}
__device__ bool is_neighbor(VX3_Voxel *voxel1, VX3_Voxel *voxel2, VX3_Link *incoming_link, int depth) {
    // printf("Checking (%d,%d,%d) and (%d,%d,%d) in depth %d.\n",
    //             voxel1->ix, voxel1->iy, voxel1->iz,
    //             voxel2->ix, voxel2->iy, voxel2->iz, depth);
    if (voxel1 == voxel2) {
        // printf("found.\n");
        return true;
    }
    if (depth <= 0) { // cannot find in depth
        // printf("not found.\n");
        return false;
    }
    for (int i = 0; i < 6; i++) {
        if (voxel1->links[i]) {
            if (voxel1->links[i] != incoming_link) {
                if (voxel1->links[i]->pVNeg == voxel1) {
                    if (is_neighbor(voxel1->links[i]->pVPos, voxel2, voxel1->links[i], depth - 1)) {
                        return true;
                    }
                } else {
                    if (is_neighbor(voxel1->links[i]->pVNeg, voxel2, voxel1->links[i], depth - 1)) {
                        return true;
                    }
                }
            }
        }
    }
    // printf("not found.\n");
    return false;
}

__device__ void handle_collision_attachment(VX3_Voxel *voxel1, VX3_Voxel *voxel2, double watchDistance, VX3_VoxelyzeKernel *k) {
    // if both of the voxels are fixed, no need to compute.
    if (voxel1->mat->fixed && voxel2->mat->fixed)
        return;

    VX3_Vec3D<double> diff = voxel1->pos - voxel2->pos;
    watchDistance = (voxel1->baseSizeAverage() + voxel2->baseSizeAverage()) * COLLISION_ENVELOPE_RADIUS;

    if (diff.x > watchDistance || diff.x < -watchDistance)
        return;
    if (diff.y > watchDistance || diff.y < -watchDistance)
        return;
    if (diff.z > watchDistance || diff.z < -watchDistance)
        return;

    if (diff.Length() > watchDistance)
        return;

    // to exclude voxels already have link between them. check in depth of
    // 1, direct neighbor ignore the collision
    if (is_neighbor(voxel1, voxel2, NULL, 1)) {
        return;
    }
    // calculate and store contact force, apply and clean in
    // VX3_Voxel::force()
    // if (voxel1->mat !=
    //     voxel2->mat) { // disable same material collision for now
    VX3_Vec3D<> cache_contactForce1, cache_contactForce2;
    if (k->EnableCollision) {
        VX3_Collision collision(voxel1, voxel2);
        collision.updateContactForce();
        cache_contactForce1 = collision.contactForce(voxel1);
        cache_contactForce2 = collision.contactForce(voxel2);
        voxel1->contactForce += cache_contactForce1;
        voxel2->contactForce += cache_contactForce2;
        if ((voxel1->mat->isTarget && !voxel2->mat->isTarget) || (voxel2->mat->isTarget && !voxel1->mat->isTarget)) {
            atomicAdd(&k->collisionCount, 1);
            if (k->EnableSignals) {
                if (voxel1->mat->isTarget) {
                    voxel2->receiveSignal(100, k->currentTime, true);
                } else {
                    voxel1->receiveSignal(100, k->currentTime, true);
                }
            }
        }
    }

    // determined by formula
    if (!voxel1->enableAttach || !voxel2->enableAttach)
        return;

    // fixed voxels, no need to look further for attachment
    if (voxel1->mat->fixed || voxel2->mat->fixed)
        return;
    // different material, no need to attach
    if (voxel1->mat != voxel2->mat)
        return;
    if (!voxel1->mat->sticky)
        return;

    // to exclude voxels already have link between them. check in depth 5.
    // closely connected part ignore the link creation.
    if (is_neighbor(voxel1, voxel2, NULL, 5)) {
        return;
    }

    // determine relative position
    linkDirection link_dir_1, link_dir_2;
    linkAxis link_axis;
    auto a = voxel1->orientation();
    auto b = voxel2->orientation();
    auto c = voxel1->position();
    auto d = voxel2->position();
    auto e = c - d;
    auto ea = a.RotateVec3DInv(-e);
    auto eb = b.RotateVec3DInv(e);

    // first find which is the dominant axis, then determine which one is
    // neg which one is pos.
    VX3_Vec3D<double> f;
    bool reverseOrder = false;
    f = ea.Abs();
    if (f.x >= f.y && f.x >= f.z) { // X_AXIS
        link_axis = X_AXIS;
        if (ea.x < 0) {
            link_dir_1 = X_NEG;
            link_dir_2 = X_POS;
            reverseOrder = true;
        } else {
            link_dir_1 = X_POS;
            link_dir_2 = X_NEG;
        }
    } else if (f.y >= f.x && f.y >= f.z) { // Y_AXIS
        link_axis = Y_AXIS;
        if (ea.y < 0) {
            link_dir_1 = Y_NEG;
            link_dir_2 = Y_POS;
            reverseOrder = true;
        } else {
            link_dir_1 = Y_POS;
            link_dir_2 = Y_NEG;
        }
    } else { // Z_AXIS
        link_axis = Z_AXIS;
        if (ea.z < 0) { // voxel1 is on top
            link_dir_1 = Z_NEG;
            link_dir_2 = Z_POS;
            reverseOrder = true;
        } else {
            link_dir_1 = Z_POS;
            link_dir_2 = Z_NEG;
        }
    }

    // TODO: need to solve this. Create only when there's a right place to
    // attach
    if (voxel1->links[link_dir_1] == NULL && voxel2->links[link_dir_2] == NULL) {
        VX3_Link *pL;
        if (reverseOrder) {
            pL = new VX3_Link(voxel1, link_dir_1, voxel2, link_dir_2, link_axis,
                              k); // make the new link (change to both materials, etc.
        } else {
            pL = new VX3_Link(voxel2, link_dir_2, voxel1, link_dir_1, link_axis,
                              k); // make the new link (change to both materials, etc.
        }
        if (!pL) {
            printf(COLORCODE_BOLD_RED "ERROR: Out of memory. Link not created.\n");
            return;
        }
        pL->isNewLink = k->SafetyGuard;
        k->d_v_links.push_back(pL); // add to the list

        k->isSurfaceChanged = true;

        // printf("createLink.... %p %p distance=> %f %f %f (%f), dir (%d and "
        //        "%d), watchDistance %f.\n",
        //        voxel1, voxel2, diff.x, diff.y, diff.z, diff.Length(),
        //        link_dir_1, link_dir_2, watchDistance);
        // printf("orientation (%f; %f, %f, %f) and (%f; %f, %f, %f).\n", a.w,
        //        a.x, a.y, a.z, b.w, b.x, b.y, b.z);
        // printf("ea, after inv rotate (%f, %f, %f)", ea.x, ea.y, ea.z);
        // printf("newLink: rest %f.\n", pL->currentRestLength);
        // printf("between (%d,%d,%d) and (%d,%d,%d).\n", voxel1->ix,
        //        voxel1->iy, voxel1->iz, voxel2->ix, voxel2->iy, voxel2->iz);

        // if a link is created, set contact force = 0 , for stable reason. (if they are connected, they should not collide.)
        voxel1->contactForce -= cache_contactForce1;
        voxel2->contactForce -= cache_contactForce2;
    }
}

__global__ void gpu_update_attach(VX3_Voxel **surface_voxels, int num, double watchDistance, VX3_VoxelyzeKernel *k) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int second = threadIdx.y + blockIdx.y * blockDim.y;
    if (first < num && second < first) {
        VX3_Voxel *voxel1 = surface_voxels[first];
        VX3_Voxel *voxel2 = surface_voxels[second];
        if (voxel1->removed || voxel2->removed)
            return;
        handle_collision_attachment(voxel1, voxel2, watchDistance, k);
    }
}

// TODO: only need to update after attachment changes.
__global__ void gpu_update_cilia_force(VX3_Voxel **surface_voxels, int num, VX3_VoxelyzeKernel *k) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        if (surface_voxels[index]->removed)
            return;
        if (surface_voxels[index]->mat->Cilia == 0)
            return;
        if (surface_voxels[index]->mat->TurnOnCiliaAfterThisManySeconds > k->currentTime)
            return;
        // rotate base cilia force and update it into voxel.
        surface_voxels[index]->CiliaForce = surface_voxels[index]->orient.RotateVec3D(
            surface_voxels[index]->baseCiliaForce + surface_voxels[index]->localSignal * surface_voxels[index]->shiftCiliaForce);
    }
}

__global__ void gpu_clear_lookupgrid(VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid, int num) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        d_collisionLookupGrid[index].clear();
    }
}

__global__ void gpu_insert_lookupgrid(VX3_Voxel **d_surface_voxels, int num, VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid,
                                      VX3_Vec3D<> *gridLowerBound, VX3_Vec3D<> *gridDelta, int lookupGrid_n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        VX3_Voxel *v = d_surface_voxels[index];
        int ix = int((v->pos.x - gridLowerBound->x) / gridDelta->x);
        int iy = int((v->pos.y - gridLowerBound->y) / gridDelta->y);
        int iz = int((v->pos.z - gridLowerBound->z) / gridDelta->z);
        bound(ix, 0, lookupGrid_n);
        bound(iy, 0, lookupGrid_n);
        bound(iz, 0, lookupGrid_n);
        d_collisionLookupGrid[ix * lookupGrid_n * lookupGrid_n + iy * lookupGrid_n + iz].push_back(v);
    }
}

__global__ void gpu_pairwise_detection(VX3_Voxel **voxel1, VX3_Voxel **voxel2, int num_v1, int num_v2, double watchDistance,
                                       VX3_VoxelyzeKernel *k) {
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    if (index_x < num_v1 && index_y < num_v2) {
        if (voxel1[index_x]->removed || voxel2[index_y]->removed)
            return;
        handle_collision_attachment(voxel1[index_x], voxel2[index_y], watchDistance, k);
    }
}

__device__ int index_3d_to_1d(int x, int y, int z, int dim_len) { return x * dim_len * dim_len + y * dim_len + z; }
__device__ VX3_Vec3D<int> index_1d_to_3d(int n, int dim_len) {
    VX3_Vec3D<int> v;
    v.x = int(floor(double(n / (dim_len * dim_len)))) % dim_len;
    v.y = int(floor(double(n / dim_len))) % dim_len;
    v.z = n % dim_len;
    return v;
}

__global__ void gpu_collision_attachment_lookupgrid(VX3_dVector<VX3_Voxel *> *d_collisionLookupGrid, int num, double watchDistance,
                                                    VX3_VoxelyzeKernel *k) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < num) {
        int num_voxel_in_grid = d_collisionLookupGrid[index].size();
        if (num_voxel_in_grid == 0)
            return;
        // within the grid
        int dim_len = k->lookupGrid_n;
        auto index_3d = index_1d_to_3d(index, dim_len);
        int ix = index_3d.x;
        int iy = index_3d.y;
        int iz = index_3d.z;
        // printf("num_voxel_in_grid %d[%d][%d][%d]: %d\n", index, ix, iy, iz, num_voxel_in_grid);
        int blockSize = 16;
        dim3 dimBlock(blockSize, blockSize);
        dim3 dimGrid((num_voxel_in_grid + dimBlock.x - 1) / dimBlock.x, (num_voxel_in_grid + dimBlock.y - 1) / dimBlock.y);
        gpu_pairwise_detection<<<dimGrid, dimBlock>>>(&d_collisionLookupGrid[index][0], &d_collisionLookupGrid[index][0], num_voxel_in_grid,
                                                      num_voxel_in_grid, watchDistance, k);
        // invoke two dimensional gpu threads 'CUDA C++ Programming
        // Guide', Nov 2019, P52.
        CUDA_CHECK_AFTER_CALL();
        // with neighbors
        for (int dix = -1; dix <= 1; dix++) {
            for (int diy = -1; diy <= 1; diy++) {
                for (int diz = -1; diz <= 1; diz++) {
                    int index_2 = index_3d_to_1d(ix + dix, iy + diy, iz + diz, dim_len);
                    if (index_2 > index && index_2 < num) {
                        int num_voxel_in_grid_2 = d_collisionLookupGrid[index_2].size();
                        if (num_voxel_in_grid_2 > 0) {
                            gpu_pairwise_detection<<<dimGrid, dimBlock>>>(
                                &d_collisionLookupGrid[index][0],
                                &d_collisionLookupGrid[index_3d_to_1d(ix + dix, iy + diy, iz + diz, dim_len)][0], num_voxel_in_grid,
                                num_voxel_in_grid_2, watchDistance, k);
                        }
                    }
                }
            }
        }
        CUDA_CHECK_AFTER_CALL();
    }
}

__global__ void gpu_update_detach(VX3_Link **links, int num, VX3_VoxelyzeKernel* k) {
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < num) {
        VX3_Link *t = links[gindex];
        if (t->removed)
            return;
        if (t->isDetached)
            return;
        // clu: vxa: MatModel=1, Fail_Stress=1e+6 => Fail_Stress => failureStress => isFailed.
        if (t->isFailed()) {
            t->isDetached = true;
            for (int i = 0; i < 6; i++) {
                if (t->pVNeg->links[i] == t) {
                    t->pVNeg->links[i] = NULL;
                }
                if (t->pVPos->links[i] == t) {
                    t->pVPos->links[i] = NULL;
                }
            }
            k->isSurfaceChanged = true;
        }
    }
}