#include "hip/hip_runtime.h"
#include "hip/hip_math_constants.h"

#include "VX3_Link.h"
#include "VX3_MaterialLink.h"
#include "VX3_VoxelyzeKernel.cuh"
#include <vector>

VX3_Link::VX3_Link(CVX_Link *p, VX3_VoxelyzeKernel *k)
    : forceNeg(p->forceNeg), forcePos(p->forcePos), momentNeg(p->momentNeg), momentPos(p->momentPos), strain(p->strain),
      maxStrain(p->maxStrain), strainOffset(p->strainOffset), boolStates(p->boolStates), strainRatio(p->strainRatio),
      pos2(p->pos2), angle1v(p->angle1v), angle2v(p->angle2v), angle1(p->angle1), angle2(p->angle2), smallAngle(p->smallAngle),
      currentRestLength(p->currentRestLength), currentTransverseArea(p->currentTransverseArea),
      currentTransverseStrainSum(p->currentTransverseStrainSum), _stress(p->_stress) {
    pVNeg = k->h_lookup_voxels[p->pVNeg];
    pVPos = k->h_lookup_voxels[p->pVPos];
    for (int i = 0; i < k->num_d_linkMats; i++) {
        if (k->h_linkMats[i] == p->mat) {
            mat = &k->d_linkMats[i];
        }
    }
}
// VX3_Link can also be initialized in device
__device__ VX3_Link::VX3_Link(VX3_Voxel *voxelNeg, linkDirection dirNeg, VX3_Voxel *voxelPos, linkDirection dirPos, VX3_VoxelyzeKernel *k) {
    deviceInit(k);
    voxelNeg->links[dirNeg] = this;
    voxelPos->links[dirPos] = this;
    pVNeg = voxelNeg;
    pVPos = voxelPos;
    linkdirNeg = dirNeg;
    linkdirPos = dirPos;
    
    mat = k->combinedMaterial(voxelNeg->material(), voxelPos->material());
    boolStates = 0;

    reset();
}

__device__ void VX3_Link::deviceInit(VX3_VoxelyzeKernel *k) {
    d_kernel = k;

    double num71 = cos(HIP_PIO4);
    quat_linkDirection[(int)X_POS] = VX3_Quat3D<double>(1,0,0,0);
    quat_linkDirection[(int)X_NEG] = VX3_Quat3D<double>(0,0,0,-1);
    quat_linkDirection[(int)Y_POS] = VX3_Quat3D<double>(num71,0,0,num71);
    quat_linkDirection[(int)Y_NEG] = VX3_Quat3D<double>(num71,0,0,-num71);
    quat_linkDirection[(int)Z_POS] = VX3_Quat3D<double>(num71,0,-num71,0);
    quat_linkDirection[(int)Z_NEG] = VX3_Quat3D<double>(num71,0,num71,0);
}

__device__ void VX3_Link::reset() {
    pos2 = angle1v = angle2v = VX3_Vec3D<double>();
    angle1 = angle2 = VX3_Quat3D<double>();
    forceNeg = forcePos = momentNeg = momentPos = VX3_Vec3D<double>();
    strain = maxStrain = strainOffset = _stress = 0.0f;
    strainRatio = pVPos->material()->E / pVNeg->material()->E;
    smallAngle = true;

    setBoolState(LOCAL_VELOCITY_VALID, false);

    updateRestLength();
    updateTransverseInfo();
}

__device__ float VX3_Link::axialStrain(bool positiveEnd) const {
    return positiveEnd ? 2.0f * strain * strainRatio / (1.0f + strainRatio) : 2.0f * strain / (1.0f + strainRatio);
}

__device__ bool VX3_Link::isYielded() const { return mat->isYielded(maxStrain); }

__device__ bool VX3_Link::isFailed() const {
    
    // sam: If this tag is zero then just see if the link exceeds the maxStrain.
    //      The link will break isFailed and MatModel == 1 (MDL_LINEAR_FAIL).
    if (mat->FailStressAddedStrengthPerNeighbor == 0.0f) 
        return mat->isFailed(maxStrain);

    // sam:
    // count the neighbors of the two voxels connected by this link
    float numNeighbors = 1;
    for (int i = 0; i < 6; i++) {
        if (pVNeg->links[i] && (pVNeg->links[i] != this)) numNeighbors += 1;
        if (pVPos->links[i] && (pVPos->links[i] != this)) numNeighbors += 1;
    }
    // beam strength is proportional to number of neighbors:
    float additionalStrength = numNeighbors * mat->FailStressAddedStrengthPerNeighbor;
    return mat->isFailed(maxStrain, additionalStrength);  // strain>epsilonFail*additionalStrength
}

__device__ void VX3_Link::updateRestLength() {
    // update rest length according to temperature of both end
    currentRestLength = 0.5 * (pVNeg->baseSize(toAxis(linkdirNeg)) + pVPos->baseSize(toAxis(linkdirPos)));
}

__device__ void VX3_Link::updateTransverseInfo() {
    currentTransverseArea = 0.5f * (pVNeg->transverseArea(toAxis(linkdirNeg)) + pVPos->transverseArea(toAxis(linkdirPos)));
    currentTransverseStrainSum = 0.5f * (pVNeg->transverseStrainSum(toAxis(linkdirNeg)) + pVPos->transverseStrainSum(toAxis(linkdirPos)));
}

/*
    Important Notes [Sida's Best Guess]:

    refer to: VX3_Link_orientLink.jpg

    In local coordinates, the link is always in the X_POS direction. pVNeg--->pVPos

    raw_pos2: position of pVPos relative to pVNeg in raw coordinates.
    local_pos2: position of pVPos relative to pVNeg in local coordinates.
    raw_angle1: orientation of pVNeg in raw coordinates.
    raw_angle2: orientation of pVPos in raw coordinates.
    totalRotation: a Quat rotate from real (raw) coordinates to local coordinates.
    pos2: position of pVPos in local corrdinates.
    angle1: pVNeg's orientation in local coordinates.
    angle2: pVPos's orientation in local coordinates
    final pos2: position of pVPos relative to rest place in local coordinates.

    About Quat3D=sin(theta/2)+cos(theta/2)(ijk-vector): When the observer look from ijk-vector to the origin, the rotation is counter-clockwise angle theta.
 */
__device__ void VX3_Link::orientLink() // updates pos2, angle1, angle2, and smallAngle
{
    if (true) {
        // New method, using quant_linkDirection.
        // 1. pVNeg->orientation
        // 2. linkdirNeg
        // 1+2 => 3. totalRotation
        // 4. raw pos2
        // 5. pVPos->orientation
        // 6. linkdirPos
        // 5+6 => 7. raw angle2
        // 3+4 => 8. pos2
        // 3+7 => 9. angle2
        // Imagine that pVNeg is placed on the origin, and the linkdirNeg is pointing towards +X direction.

        VX3_Quat3D<> r1 = pVNeg->orientation().Conjugate(); // (1.) Rotate things from real coordinates into pVNeg's oritentation
        VX3_Quat3D<> r2 = quat_linkDirection[linkdirNeg].Conjugate(); // (2.) Rotate things from pVNeg's orientation into linkdirNeg's orientation
        VX3_Quat3D<> r3 = quat_linkDirection[oppositeDirection(linkdirPos)]; // (6.) Rotate things from the opposite of linkdirPos's orientation back to pVPos's orientation
        VX3_Quat3D<> totalRotation = r2*r1; //(3.)
        VX3_Vec3D<> raw_pos2 = pVPos->position() - pVNeg->position(); //(4.)
        pos2 = totalRotation.RotateVec3D(raw_pos2); //(8.)
        angle2 = totalRotation * pVPos->orientation() * r3; // (9.)
        angle1=  VX3_Quat3D<>(); // always zero
    }
    if (false) {
        // old method
        VX3_Vec3D<> _pos2 = pVPos->position() - pVNeg->position();
        pos2 = toAxisX(_pos2, toAxis(linkdirNeg)); // digit truncation happens here...
        VX3_Quat3D<> _angle1 = pVNeg->orientation();
        angle1 = toAxisX(_angle1, toAxis(linkdirNeg));
        VX3_Quat3D<> _angle2 = pVPos->orientation();
        angle2 = toAxisX(_angle2, toAxis((linkDirection)oppositeDirection(linkdirPos)));

        VX3_Quat3D<double> totalRot = angle1.Conjugate(); // keep track of the total rotation of this bond
                                                        // (after toAxisX())
        pos2 = totalRot.RotateVec3D(pos2);
        angle2 = totalRot * angle2;
        angle1 = VX3_Quat3D<>(); // zero for now...         
    }

    // small angle approximation?
    float SmallTurn = (float)((abs(pos2.z) + abs(pos2.y)) / pos2.x);
    float ExtendPerc = (float)(abs(1 - pos2.x / currentRestLength));
    if (!smallAngle /*&& angle2.IsSmallAngle()*/ && SmallTurn < SA_BOND_BEND_RAD && ExtendPerc < SA_BOND_EXT_PERC) {
        smallAngle = true;
        setBoolState(LOCAL_VELOCITY_VALID, false);
    } else if (smallAngle && (/*!angle2.IsSmallishAngle() || */ SmallTurn > HYSTERESIS_FACTOR * SA_BOND_BEND_RAD ||
                              ExtendPerc > HYSTERESIS_FACTOR * SA_BOND_EXT_PERC)) {
        smallAngle = false;
        setBoolState(LOCAL_VELOCITY_VALID, false);
    }

    //small angle means the link is stabled. wait until SafetyGuard(isNewLink) decreased to 0.
    if (smallAngle && isNewLink>0) {
        isNewLink--;
        if (isNewLink==0) {
            // Great, this link is stablized, remove one from the whole group.
            pVNeg->d_group->hasNewLink--;
        }
    }

    // is small angle, we are using ideal X_POS direction as local direction
    // otherwise, we are using the direction from the center of mass of pVNeg to pVPos. Then, angle1 is not 0 anymore.
    if (smallAngle) {                 // Align so Angle1 is all zeros
        pos2.x -= currentRestLength;  // only valid for small angles
    } else {                          // Large angle. Align so that Pos2.y, Pos2.z are zero.
        angle1.FromAngleToPosX(pos2); // get the angle to align Pos2 with the X axis
        angle2 = angle1 * angle2;     // rotate angle2
        pos2 = VX3_Vec3D<>(pos2.Length() - currentRestLength, 0, 0);
    }
    // angle1.Rotate() can rotate everything from imaginary perfect beam corrdinates to real beam coordinates.
    // angle2.Rotate() can rotate everything from real coordinates to real beam coordinates.
    // Roughly speaking, angle1 is pVNeg's rotation in beam coordinates, angle2 is pVPos's, ignoring the linkDirection.
    angle1v = angle1.ToRotationVector();
    angle2v = angle2.ToRotationVector();

    assert(!(angle1v.x != angle1v.x) || !(angle1v.y != angle1v.y) || !(angle1v.z != angle1v.z)); // assert non QNAN
    assert(!(angle2v.x != angle2v.x) || !(angle2v.y != angle2v.y) || !(angle2v.z != angle2v.z)); // assert non QNAN
}

__device__ void VX3_Link::updateForces() {
    VX3_Vec3D<double> oldPos2 = pos2;
    VX3_Vec3D<double> oldAngle1v = angle1v;
    VX3_Vec3D<double> oldAngle2v = angle2v; // remember the positions/angles from last timestep to
                                            // calculate velocity

    orientLink();                                     // sets pos2, angle1, angle2
    VX3_Vec3D<double> dPos2 = 0.5 * (pos2 - oldPos2); // deltas for local damping. velocity at center
                                                      // is half the total velocity
    VX3_Vec3D<double> dAngle1 = 0.5 * (angle1v - oldAngle1v);
    VX3_Vec3D<double> dAngle2 = 0.5 * (angle2v - oldAngle2v);
    // if volume effects..
    if (!mat->isXyzIndependent() || currentTransverseStrainSum != 0) { // currentTransverseStrainSum != 0 catches when we disable
                                                                       // poissons mid-simulation
        updateTransverseInfo();
    }
    _stress = updateStrain((float)(pos2.x / currentRestLength));
    if (isFailed()) {
        forceNeg = forcePos = momentNeg = momentPos = VX3_Vec3D<double>(0, 0, 0);
        if (d_kernel->enableDetach) {
            detach();
        }
        return;
    }
    float b1 = mat->_b1, b2 = mat->_b2, b3 = mat->_b3,
          a2 = mat->_a2; // local copies
    // Beam equations. All relevant terms are here, even though some are zero
    // for small angle and others are zero for large angle (profiled as
    // negligible performance penalty)
    forceNeg = VX3_Vec3D<double>(_stress * currentTransverseArea, // currentA1*pos2.x,
                                 b1 * pos2.y - b2 * (angle1v.z + angle2v.z),
                                 b1 * pos2.z + b2 * (angle1v.y + angle2v.y)); // Use Curstress instead of -a1*Pos2.x
                                                                              // to account for non-linear deformation
    forcePos = -forceNeg;

    momentNeg = VX3_Vec3D<double>(a2 * (angle2v.x - angle1v.x), -b2 * pos2.z - b3 * (2 * angle1v.y + angle2v.y),
                                  b2 * pos2.y - b3 * (2 * angle1v.z + angle2v.z));
    momentPos = VX3_Vec3D<double>(a2 * (angle1v.x - angle2v.x), -b2 * pos2.z - b3 * (angle1v.y + 2 * angle2v.y),
                                  b2 * pos2.y - b3 * (angle1v.z + 2 * angle2v.z));

    // local damping: (I don't understand this damping calculation. wouldn't damping_force = -velocity * coefficient easier?)
    if (isLocalVelocityValid()) { // if we don't have the basis for a good
                                  // damping calculation, don't do any damping.
        float neg_damp = pVNeg->dampingMultiplier();
        float pos_damp = pVPos->dampingMultiplier();

        float sqA1 = mat->_sqA1, sqA2xIp = mat->_sqA2xIp, sqB1 = mat->_sqB1, sqB2xFMp = mat->_sqB2xFMp, sqB3xIp = mat->_sqB3xIp;
        VX3_Vec3D<double> posCalc(sqA1 * dPos2.x, sqB1 * dPos2.y - sqB2xFMp * (dAngle1.z + dAngle2.z),
                                  sqB1 * dPos2.z + sqB2xFMp * (dAngle1.y + dAngle2.y));

        forceNeg += neg_damp * posCalc;
        forcePos -= pos_damp * posCalc;

        momentNeg -= 0.5 * neg_damp *
                     VX3_Vec3D<>(-sqA2xIp * (dAngle2.x - dAngle1.x), sqB2xFMp * dPos2.z + sqB3xIp * (2 * dAngle1.y + dAngle2.y),
                                 -sqB2xFMp * dPos2.y + sqB3xIp * (2 * dAngle1.z + dAngle2.z));
        momentPos -= 0.5 * pos_damp *
                     VX3_Vec3D<>(sqA2xIp * (dAngle2.x - dAngle1.x), sqB2xFMp * dPos2.z + sqB3xIp * (dAngle1.y + 2 * dAngle2.y),
                                 -sqB2xFMp * dPos2.y + sqB3xIp * (dAngle1.z + 2 * dAngle2.z));
                               
    } else
        setBoolState(LOCAL_VELOCITY_VALID,
                     true); // we're good for next go-around unless something changes

    //	transform forces and moments to local voxel coordinates
    if (!smallAngle) {
        forceNeg = angle1.RotateVec3DInv(forceNeg);
        momentNeg = angle1.RotateVec3DInv(momentNeg);
    }
    forcePos = angle2.RotateVec3DInv(forcePos);
    momentPos = angle2.RotateVec3DInv(momentPos);

    // Rewrite Rotation back, so linkNeg and linkPos don't need to be paired. (For arbitrary attachment.)
    if (true) {
        // new method
        forceNeg = quat_linkDirection[linkdirNeg].RotateVec3D(forceNeg);
        forcePos = quat_linkDirection[linkdirNeg].RotateVec3D(forcePos);
        momentNeg = quat_linkDirection[linkdirNeg].RotateVec3D(momentNeg);
        momentPos = quat_linkDirection[linkdirNeg].RotateVec3D(momentPos);
    } 
    if (false) {
        // old method
        toAxisOriginal(&forceNeg, toAxis(linkdirNeg));
        toAxisOriginal(&forcePos, toAxis(linkdirNeg));
        toAxisOriginal(&momentNeg, toAxis(linkdirNeg));
        toAxisOriginal(&momentPos, toAxis(linkdirNeg));
    }

    assert(!(forceNeg.x != forceNeg.x) || !(forceNeg.y != forceNeg.y) || !(forceNeg.z != forceNeg.z)); //assert non QNAN
    assert(!(forcePos.x != forcePos.x) || !(forcePos.y != forcePos.y) || !(forcePos.z != forcePos.z)); //assert non QNAN
}

__device__ float VX3_Link::updateStrain(float axialStrain) {
    if (mat->linear) {

        if (axialStrain > maxStrain)
            maxStrain = axialStrain; // remember this maximum for easy reference

        return mat->stress(axialStrain, currentTransverseStrainSum);
    } else {
        float returnStress;

        if (axialStrain > maxStrain) { // if new territory on the stress/strain curve
            maxStrain = axialStrain;   // remember this maximum for easy reference
            returnStress = mat->stress(axialStrain, currentTransverseStrainSum);

            if (mat->nu != 0.0f)
                strainOffset = maxStrain - mat->stress(axialStrain) / (mat->_eHat * (1 - mat->nu)); // precalculate strain offset for when
                                                                                                    // we back off
            else
                strainOffset = maxStrain - returnStress / mat->E; // precalculate strain offset for
                                                                  // when we back off

        } else { // backed off a non-linear material, therefore in linear
                 // region.

            float relativeStrain = axialStrain - strainOffset; // treat the material as linear with
                                                               // a strain offset according to the
                                                               // maximum plastic deformation

            if (mat->nu != 0.0f)
                returnStress = mat->stress(relativeStrain, currentTransverseStrainSum, true);
            else
                returnStress = mat->E * relativeStrain;
        }

        return returnStress;
    }
}

__device__ float VX3_Link::strainEnergy() const {
    return forceNeg.x * forceNeg.x / (2.0f * mat->_a1) +                                                            // Tensile strain
           momentNeg.x * momentNeg.x / (2.0 * mat->_a2) +                                                           // Torsion strain
           (momentNeg.z * momentNeg.z - momentNeg.z * momentPos.z + momentPos.z * momentPos.z) / (3.0 * mat->_b3) + // Bending Z
           (momentNeg.y * momentNeg.y - momentNeg.y * momentPos.y + momentPos.y * momentPos.y) / (3.0 * mat->_b3);  // Bending Y
}

__device__ float VX3_Link::axialStiffness() {
    if (mat->isXyzIndependent())
        return mat->_a1;
    else {
        updateRestLength();
        updateTransverseInfo();

        return (float)(mat->_eHat * currentTransverseArea / ((strain + 1) * currentRestLength)); // _a1;
    }
}

__device__ float VX3_Link::a1() const { return mat->_a1; }
__device__ float VX3_Link::a2() const { return mat->_a2; }
__device__ float VX3_Link::b1() const { return mat->_b1; }
__device__ float VX3_Link::b2() const { return mat->_b2; }
__device__ float VX3_Link::b3() const { return mat->_b3; }

__device__ void VX3_Link::detach() {
    isDetached = true;
    
    pVNeg->links[linkdirNeg] = NULL;
    pVPos->links[linkdirPos] = NULL;

    // pVPos->d_group = new VX3_VoxelGroup(d_kernel);
    pVPos->d_group = (VX3_VoxelGroup*) hamalloc(sizeof(VX3_VoxelGroup));
    if (pVPos->d_group==NULL) {
        printf("halloc: Out of memory. Please increate the size of memory that halloc manages.\n");
    }
    pVPos->d_group->deviceInit(d_kernel);
    
    PRINT(d_kernel, "detach: create a d_group = %p.\n", pVPos->d_group);
    pVPos->d_group->needUpdate = true;
    pVNeg->d_group->needUpdate = true;
    d_kernel->d_voxel_to_update_group.push_back(pVPos);
    d_kernel->d_voxel_to_update_group.push_back(pVNeg);
    pVPos->d_group->d_voxels.push_back(pVPos);
    d_kernel->d_voxelgroups.push_back(pVPos->d_group);

    d_kernel->isSurfaceChanged = true;
}
