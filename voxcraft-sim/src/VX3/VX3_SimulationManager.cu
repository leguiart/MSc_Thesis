#include "hip/hip_runtime.h"
#include "VX3_SimulationManager.cuh"
#include "ctool.h"
#include <boost/algorithm/string/case_conv.hpp>
#include <queue>
#include <stack>
#include <utility>

#include <chrono>
#include <ctime>
#include <ratio>

#include "VX3_VoxelyzeKernel.cuh"
#include "VX_Sim.h" //readVXA

__device__ void _CUDA_Simulation(VX3_VoxelyzeKernel *k, int thread_index, int device_index);

__global__ void sequential_CUDA_Simulation(VX3_VoxelyzeKernel *d_voxelyze_3, int num_simulation, int device_index) {
    for (int i = 0; i < num_simulation; i++) {
        printf("Sequentially starting Simulation %d/%d on GPU %d.\n", i, num_simulation, device_index);
        _CUDA_Simulation(&d_voxelyze_3[i], i, device_index);
    }
}
__global__ void CUDA_Simulation(VX3_VoxelyzeKernel *d_voxelyze_3, int num_simulation, int device_index) {
    int thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index < num_simulation) {
        _CUDA_Simulation(&d_voxelyze_3[thread_index], thread_index, device_index);
    }
}
__device__ void _CUDA_Simulation(VX3_VoxelyzeKernel *k, int thread_index, int device_index) {
    {
        VX3_VoxelyzeKernel *d_v3 = k;
        if (d_v3->num_d_links == 0 and d_v3->num_d_voxels == 0) {
            printf(COLORCODE_BOLD_RED "No links and no voxels. Simulation %d (%s) abort.\n" COLORCODE_RESET, thread_index, d_v3->vxa_filename);
            return;
        }
        d_v3->deviceInit(); // Everytime we pass a class with VX3_vectors in
                            // it, we should sync hd_vector to d_vector first.
        d_v3->saveInitialPosition();
        d_v3->isSurfaceChanged = true; // trigger surface regenerating and calculate normal thrust for the first time
        d_v3->registerTargets();
        printf(COLORCODE_GREEN "%d) Simulation %d runs: %s.\n" COLORCODE_RESET, device_index, thread_index, d_v3->vxa_filename);
        // printf("%d) Simulation %d: links %d, voxels %d.\n", device_index, i,
        // d_v3->num_d_links, d_v3->num_d_voxels); printf("%d) Simulation %d
        // enableAttach %d.\n", device_index, i, d_v3->enableAttach);
        //
        // print check regenerateSurfaceVoxels() is correct. (TODO: shouldn't
        // this be tested in seperate test code? :) printf("all voxels:"); for
        // (int j=0;j<d_v3->num_d_voxels;j++) {
        //     printf(" [%d]%p ", j, &d_v3->d_voxels[j]);
        // }
        // printf("\nsurface:");
        // for (int j=0;j<d_v3->num_d_surface_voxels;j++) {
        //     printf(" [%d]%p ", j, d_v3->d_surface_voxels[j]);
        // }
        //
        if (d_v3->RecordStepSize) { // output History file
            // rescale the whole space. so history file can contain less digits. ( e.g. not 0.000221, but 2.21 )
            printf("\n{{{setting}}}<rescale>0.001</rescale>\n");
            // materials' color
            for (int i = 0; i < d_v3->num_d_voxelMats; i++) {
                auto &mat = d_v3->d_voxelMats[i];
                printf("{{{setting}}}<matcolor><id>%d</id><r>%.2f</r><g>%.2f</g><b>%.2f</b><a>%.2f</a></matcolor>\n", mat.matid, mat.r / 255., mat.g / 255., mat.b / 255., mat.a / 255.);
            }
            printf("\n{{{setting}}}<voxel_size>%f</voxel_size>\n", d_v3->voxSize);
        }

        double vs = 1 / 0.001;

        d_v3->updateCurrentCenterOfMass();
        d_v3->InitializeCenterOfMass();
        int real_stepsize = int(d_v3->RecordStepSize / (10000 * d_v3->recommendedTimeStep() * d_v3->DtFrac)) + 1;
        printf("real_stepsize: %d ; recommendedTimeStep %f; d_v3->DtFrac %f . \n", real_stepsize, d_v3->recommendedTimeStep(), d_v3->DtFrac);
        // printf("Initial CoM: %f %f %f mm\n",
        // d_v3->initialCenterOfMass.x*1000, d_v3->initialCenterOfMass.y*1000,
        // d_v3->initialCenterOfMass.z*1000);
        for (int j = 0; j < 1000000; j++) { // Maximum Steps 1000000
            // if (true || j%10 ==0) {
            //     int total_valid_groups = 0;
            //     for (int _i = 0;_i<d_v3->d_voxelgroups.size();_i++) {
            //         if (!d_v3->d_voxelgroups[_i]->removed)
            //             total_valid_groups ++;
            //     }
            //     PRINT(d_v3, "Number of groups: %d \n", total_valid_groups);
            // }

            if (d_v3->StopConditionMet()) {
                printf("Stopping after %d timesteps\n", j);
                break;
            }

            // sam: for secondary experiment
            if (d_v3->EarlyStopIfNoBotsRemain()) {
                printf("No bots left with minimal size %d. Stopping after %d timesteps\n", d_v3->MinimumBotSize, j);
                break;
            }

            if (!d_v3->doTimeStep()) {
                printf(COLORCODE_BOLD_RED "\n%d) Simulation %d Diverged: %s.\n" COLORCODE_RESET, device_index, thread_index, d_v3->vxa_filename);
                break;
            }
            if (d_v3->RecordStepSize) { // output History file
                if (j % real_stepsize == 0) {
                    // { // debug, print all the voxels and groups.
                    //     printf("=========== Step %d ===========\n", j);
                    //     for (int k=0;k<d_v3->d_voxelgroups.size();k++) {
                    //         VX3_VoxelGroup* tmp = d_v3->d_voxelgroups[k];
                    //         if (!tmp->removed) {
                    //             printf("Group: %d (%p). \n\tremoved: %d, dim: (%d, %d, %d).\n", k, tmp, (int)tmp->removed, tmp->dim_x, tmp->dim_y, tmp->dim_z);
                    //         }
                    //     }
                    //     continue;
                    //     for (int k=0;k<d_v3->num_d_voxels;k++) {
                    //         VX3_Voxel* tmp = &d_v3->d_voxels[k];
                    //         printf("Voxel: %d (%p). \n\t removed: %d, ix: %d, iy: %d, iz: %d.\n", k, tmp, (int)tmp->removed, tmp->ix, tmp->iy, tmp->iz);
                    //     }
                    // }

                    if (d_v3->RecordVoxel) {
                        // Voxels
                        printf("<<<Step%d Time:%f>>>", j, d_v3->currentTime);
                        int num = d_v3->num_d_voxels;
                        if (d_v3->SurfaceVoxelsOnly) {
                            num = d_v3->num_d_surface_voxels;
                        }
                        VX3_Voxel *v;
                        for (int i = 0; i < num; i++) {
                            if (d_v3->SurfaceVoxelsOnly) {
                                v = d_v3->d_surface_voxels[i];
                            } else {
                                v = &d_v3->d_voxels[i];
                            }
                            if (v->removed)
                                continue;

                            // sam:
                            if ((!d_v3->RecordFixedVoxels) && (v->mat->fixed))
                                continue;

                            //    if (v->isSurface()) {
                            printf("%.1f,%.1f,%.1f,", v->pos.x * vs, v->pos.y * vs, v->pos.z * vs);
                            printf("%.1f,%.2f,%.2f,%.2f,", v->orient.AngleDegrees(), v->orient.x, v->orient.y, v->orient.z);
                            VX3_Vec3D<double> ppp, nnn;
                            nnn = v->cornerOffset(NNN);
                            ppp = v->cornerOffset(PPP);
                            printf("%.1f,%.1f,%.1f,%.1f,%.1f,%.1f,", nnn.x * vs, nnn.y * vs, nnn.z * vs, ppp.x * vs, ppp.y * vs, ppp.z * vs);
                            printf("%d,", v->mat->matid);    // for coloring
                            printf("%.1f,", v->localSignal); // for coloring as well.
                            printf(";");
                            //    }
                        }
                        printf("<<<>>>");
                    }
                    if (d_v3->RecordLink) {
                        // Links
                        printf("|[[[%d]]]", j);
                        for (int i = 0; i < d_v3->d_v_links.size(); i++) {
                            auto l = d_v3->d_v_links[i];
                            if (l->removed)
                                continue;

                            // sam:
                            if (!d_v3->RecordFixedVoxels) {
                                if (l->pVNeg->mat->fixed || l->pVPos->mat->fixed)
                                    continue;
                            }

                            // sam:
                            if (d_v3->SurfaceVoxelsOnly) {
                                if (l->pVNeg->isInterior() || l->pVPos->isInterior())
                                    continue;
                            }

                            // only draw links that are not detached.
                            if (!l->isDetached) {
                                auto v1 = l->pVPos;
                                printf("%.4f,%.4f,%.4f,", v1->pos.x, v1->pos.y, v1->pos.z);
                                auto v2 = l->pVNeg;
                                printf("%.4f,%.4f,%.4f,", v2->pos.x, v2->pos.y, v2->pos.z);
                                printf(";");
                            }
                        }
                        printf("[[[]]]");
                    }
                    if (d_v3->RecordCoMTraceOfEachVoxelGroupfOfThisMaterial) {
                        if (j % d_v3->RecordCoMTraceOfEachVoxelGroupfOfThisMaterial == 0) {
                            printf("|{{{%d}}}", j);
                            int num_groups = d_v3->d_voxelgroups.size();
                            VX3_dDictionary<VX3_VoxelGroup *, int> visited_groups;
                            for (int CoMi = 0; CoMi < d_v3->num_d_voxels; CoMi++) {
                                if (d_v3->d_voxels[CoMi].removed) {
                                    continue;
                                }

                                // sam:
                                if (d_v3->d_voxels[CoMi].matid != d_v3->RecordCoMTraceOfEachVoxelGroupfOfThisMaterial)
                                    continue;

                                VX3_VoxelGroup *g = d_v3->d_voxels[CoMi].d_group;
                                if (visited_groups.get(g) == 1) {
                                    continue;
                                }
                                visited_groups.set(g, 1);

                                VX3_Vec3D<double> TotalPosition;
                                int num_voxels = g->d_voxels.size();
                                for (int id_voxel = 0; id_voxel < num_voxels; id_voxel++) {
                                    if (g->d_voxels[id_voxel]->removed) {
                                        continue;
                                    }
                                    TotalPosition += g->d_voxels[id_voxel]->pos;
                                }
                                TotalPosition /= num_voxels;
                                printf("%.4f,%.4f,%.4f,", TotalPosition.x, TotalPosition.y, TotalPosition.z);
                                printf(";");
                            }
                            printf("{{{}}}");
                        }
                    }
                    printf("\n");
                }
            }
        }
        d_v3->updateCurrentCenterOfMass();
        d_v3->computeFitness();
        // d_v3->computeTargetCloseness();
        // d_v3->computeNumRealLinks();
        printf(COLORCODE_BLUE "%d) Simulation %d ends: %s Time: %f, PileSize: %d.\n" COLORCODE_RESET, device_index, thread_index, d_v3->vxa_filename, d_v3->currentTime, d_v3->largestStickyGroupSize); // sam
    }
}

VX3_SimulationManager::VX3_SimulationManager(std::vector<std::vector<fs::path>> in_sub_batches, fs::path in_base, fs::path in_input_dir, int in_num_of_devices) : sub_batches(in_sub_batches), base(in_base), num_of_devices(in_num_of_devices), input_dir(in_input_dir) {
    d_voxelyze_3s.resize(num_of_devices);
    for (int i = 0; i < num_of_devices; i++) {
        d_voxelyze_3s[i] = NULL;
    }
}
VX3_SimulationManager::~VX3_SimulationManager() {
    for (auto d : d_voxelyze_3s) {
        if (d)
            VcudaFree(d);
    }
}

void VX3_SimulationManager::start() {

    using namespace std::chrono;
    high_resolution_clock::time_point t1 = high_resolution_clock::now();

    for (int device_index = 0; device_index < num_of_devices; device_index++) { // multi GPUs
        auto files = sub_batches[device_index];
        if (files.size()) {
            VcudaSetDevice(device_index);
            // Initialize halloc to manage device memory
            // ha_init(halloc_opts_t());
            ha_init(halloc_opts_t((size_t)1024 * 1024 * 1024)); // sam

            hipDeviceSetLimit(hipLimitPrintfFifoSize, 1 << 26);
            printf("=== set device to %d for %ld simulations ===\n", device_index, files.size());
            // readVXA(base)
            readVXD(base, files, device_index);
            startKernel(files.size(), device_index);
        }
    }
    VcudaDeviceSynchronize();
    for (int device_index = 0; device_index < num_of_devices; device_index++) { // multi GPUs
        auto files = sub_batches[device_index];
        collectResults(files.size(), device_index);
    }
    sortResults();

    high_resolution_clock::time_point t2 = high_resolution_clock::now();

    duration<double> time_span = duration_cast<duration<double>>(t2 - t1);
    printf("Simulation took %.4f seconds.\n", time_span.count());
}

void VX3_SimulationManager::ParseMathTree(VX3_MathTreeToken *field_ptr, size_t max_length, std::string node_address, pt::ptree &tree) {
    // Classic BFS, push all token into stack
    std::queue<pt::ptree> frontier;
    std::stack<std::pair<std::string, std::string>> tokens;
    tokens.push(make_pair((std::string) "mtEND", (std::string) ""));
    auto root = tree.get_child_optional(node_address);
    if (!root) {
        // printf(COLORCODE_BOLD_RED "ERROR: No ParseMathTree %s in VXA.\n", node_address.c_str());
        return;
    }
    frontier.push(tree.get_child(node_address));
    while (!frontier.empty()) {
        std::queue<pt::ptree> next_frontier;
        auto t = frontier.front();
        frontier.pop();
        BOOST_FOREACH (pt::ptree::value_type &v_child, t.get_child("")) {
            std::string value = v_child.second.data();
            boost::trim_right(value);
            std::string op = v_child.first.data();
            boost::trim_right(op);

            // std::cout << op << ":" << value << "\n";
            tokens.push(make_pair(op, value));
            frontier.push(v_child.second);
        }
    }
    // pop from stack to VX3_MathTreeToken* (so we get a reversed order)
    int i = 0;
    while (!tokens.empty()) {
        if (i > max_length) {
            printf(COLORCODE_BOLD_RED "ERROR: Token size overflow.\n");
            return;
        }
        std::pair<std::string, std::string> tok = tokens.top();
        VX3_MathTreeToken *p = &field_ptr[i];
        if (tok.first == "mtEND") {
            p->op = mtEND;
        } else if (tok.first == "mtVAR") {
            p->op = mtVAR;
            if (tok.second == "x") {
                p->value = 0;
            } else if (tok.second == "y") {
                p->value = 1;
            } else if (tok.second == "z") {
                p->value = 2;
            } else if (tok.second == "hit") {
                p->value = 3;
            } else if (tok.second == "t") {
                p->value = 4;
            } else if (tok.second == "angle") {
                p->value = 5;
            } else if (tok.second == "targetCloseness") {
                p->value = 6;
            } else if (tok.second == "numClosePairs") {
                p->value = 7;
            } else if (tok.second == "num_voxel") {
                p->value = 8;
            } else {
                printf(COLORCODE_BOLD_RED "ERROR: No such variable.\n");
                break;
            }
        } else if (tok.first == "mtCONST") {
            p->op = mtCONST;
            try {
                p->value = std::stod(tok.second);
            } catch (...) {
                printf(COLORCODE_BOLD_RED "ERROR: mtCONST with no number.\n");
                break;
            }
        } else if (tok.first == "mtADD") {
            p->op = mtADD;
        } else if (tok.first == "mtSUB") {
            p->op = mtSUB;
        } else if (tok.first == "mtMUL") {
            p->op = mtMUL;
        } else if (tok.first == "mtDIV") {
            p->op = mtDIV;
        } else if (tok.first == "mtPOW") {
            p->op = mtPOW;
        } else if (tok.first == "mtSQRT") {
            p->op = mtSQRT;
        } else if (tok.first == "mtE") {
            p->op = mtE;
        } else if (tok.first == "mtPI") {
            p->op = mtPI;
        } else if (tok.first == "mtSIN") {
            p->op = mtSIN;
        } else if (tok.first == "mtCOS") {
            p->op = mtCOS;
        } else if (tok.first == "mtTAN") {
            p->op = mtTAN;
        } else if (tok.first == "mtATAN") {
            p->op = mtATAN;
        } else if (tok.first == "mtLOG") {
            p->op = mtLOG;
        } else if (tok.first == "mtINT") {
            p->op = mtINT;
        } else if (tok.first == "mtABS") {
            p->op = mtABS;
        } else if (tok.first == "mtNOT") {
            p->op = mtNOT;
        } else if (tok.first == "mtGREATERTHAN") {
            p->op = mtGREATERTHAN;
        } else if (tok.first == "mtLESSTHAN") {
            p->op = mtLESSTHAN;
        } else if (tok.first == "mtAND") {
            p->op = mtAND;
        } else if (tok.first == "mtOR") {
            p->op = mtOR;
        } else if (tok.first == "mtNORMALCDF") {
            p->op = mtNORMALCDF;
        } else {
            printf(COLORCODE_BOLD_RED "ERROR: Token Operation not implemented.\n");
            break;
        }
        i++;
        tokens.pop();
    }
}

void VX3_SimulationManager::readVXD(fs::path base, std::vector<fs::path> files, int device_index) {
    pt::ptree pt_baseVXA;
    pt::read_xml(base.string(), pt_baseVXA);

    int num_simulation = files.size();

    printf("Allocate memory for the %d kernels on GPU %d.\n", num_simulation, device_index);
    VcudaMalloc((void **)&d_voxelyze_3s[device_index], num_simulation * sizeof(VX3_VoxelyzeKernel));
    CUDA_CHECK_AFTER_CALL();
    printf("Allocated kernel set (%p).\n", d_voxelyze_3s[device_index]);

    int i = 0;
    for (auto &file : files) {
        // Read VXD file, clone base VXA, replace parts specified in VXD, send
        // to MainSim.ReadVXA to process. printf("reading %s\n",
        // (input_dir/file).c_str());
        std::cout << "Reading " << file << "\n";
        pt::ptree pt_VXD;
        pt::read_xml((input_dir / file).string(), pt_VXD);
        pt::ptree pt_merged = pt_baseVXA;
        ctool::ptree_merge(pt_VXD, pt_merged);
        std::ostringstream stream_merged;
        std::string str_merged;
        pt::write_xml(stream_merged, pt_merged);
        str_merged = stream_merged.str();
        CXML_Rip XML;
        XML.fromXMLText(&str_merged);
        CVX_Environment MainEnv;
        CVX_Sim MainSim;
        CVX_Object MainObj;
        MainEnv.pObj = &MainObj; // connect environment to object
        MainSim.pEnv = &MainEnv; // connect Simulation to envirnment
        std::string RetMessage;
        // std::cout<<str_merged;
        MainSim.ReadVXA(&XML, &RetMessage);
        MainSim.Import(NULL, NULL, &RetMessage);
        if (!RetMessage.empty()) {
            printf(COLORCODE_BOLD_RED "%s\n" COLORCODE_RESET, RetMessage.c_str());
        }
        // for (auto m:MainSim.Vx.voxelMats) {
        //     int i=0;
        //     for (auto mm:m->dependentMaterials) {
        //         printf("m:%p %d/%ld -> mm: %p\n", m, i,
        //         m->dependentMaterials.size(), mm); i++;
        //     }
        // }
        VX3_VoxelyzeKernel h_d_tmp(&MainSim);
        h_d_tmp.GPU_id = device_index;
        // More VXA settings which is new in VX3
        strcpy(h_d_tmp.vxa_filename, file.filename().c_str());

        std::string RawPrint = pt_merged.get<std::string>("VXA.RawPrint", "");
        if (RawPrint.length() > 0) {
            std::cout << RawPrint << "\n";
        }
        h_d_tmp.enableFloor = pt_merged.get<bool>("VXA.Environment.Gravity.FloorEnabled", true);

        ParseMathTree(h_d_tmp.StopConditionFormula, sizeof(h_d_tmp.StopConditionFormula), "VXA.Simulator.StopCondition.StopConditionFormula", pt_merged);
        h_d_tmp.EnableCollision = pt_merged.get<bool>("VXA.Simulator.AttachDetach.EnableCollision", true);
        h_d_tmp.CollisionMode = pt_merged.get<bool>("VXA.Simulator.AttachDetach.CollisionMode", true);
        h_d_tmp.enableAttach = pt_merged.get<bool>("VXA.Simulator.AttachDetach.EnableAttach", false);
        h_d_tmp.enableDetach = pt_merged.get<bool>("VXA.Simulator.AttachDetach.EnableDetach", false);
        h_d_tmp.ForceAttachment = pt_merged.get<bool>("VXA.Simulator.AttachDetach.ForceAttachment", false); // sam
        h_d_tmp.watchDistance = pt_merged.get<double>("VXA.Simulator.AttachDetach.watchDistance", 1.0);
        h_d_tmp.boundingRadius = pt_merged.get<double>("VXA.Simulator.AttachDetach.boundingRadius", 0.75);
        h_d_tmp.SafetyGuard = pt_merged.get<int>("VXA.Simulator.AttachDetach.SafetyGuard", 500);
        ParseMathTree(h_d_tmp.AttachCondition[0], sizeof(h_d_tmp.AttachCondition[0]), "VXA.Simulator.AttachDetach.AttachCondition.Condition_0", pt_merged);
        ParseMathTree(h_d_tmp.AttachCondition[1], sizeof(h_d_tmp.AttachCondition[1]), "VXA.Simulator.AttachDetach.AttachCondition.Condition_1", pt_merged);
        ParseMathTree(h_d_tmp.AttachCondition[2], sizeof(h_d_tmp.AttachCondition[2]), "VXA.Simulator.AttachDetach.AttachCondition.Condition_2", pt_merged);
        ParseMathTree(h_d_tmp.AttachCondition[3], sizeof(h_d_tmp.AttachCondition[3]), "VXA.Simulator.AttachDetach.AttachCondition.Condition_3", pt_merged);
        ParseMathTree(h_d_tmp.AttachCondition[4], sizeof(h_d_tmp.AttachCondition[4]), "VXA.Simulator.AttachDetach.AttachCondition.Condition_4", pt_merged);
        h_d_tmp.RecordStepSize = pt_merged.get<int>("VXA.Simulator.RecordHistory.RecordStepSize", 0);
        h_d_tmp.RecordLink = pt_merged.get<int>("VXA.Simulator.RecordHistory.RecordLink", 0);
        h_d_tmp.RecordVoxel = pt_merged.get<int>("VXA.Simulator.RecordHistory.RecordVoxel", 1);
        h_d_tmp.RecordFixedVoxels = pt_merged.get<int>("VXA.Simulator.RecordHistory.RecordFixedVoxels", 1); // sam
        h_d_tmp.SurfaceVoxelsOnly = pt_merged.get<int>("VXA.Simulator.RecordHistory.SurfaceVoxelsOnly", 1);

        h_d_tmp.RecordCoMTraceOfEachVoxelGroupfOfThisMaterial = pt_merged.get<int>("VXA.Simulator.RecordHistory.RecordCoMTraceOfEachVoxelGroupfOfThisMaterial", 0);

        ParseMathTree(h_d_tmp.fitness_function, sizeof(h_d_tmp.fitness_function), "VXA.Simulator.FitnessFunction", pt_merged);
        ParseMathTree(h_d_tmp.force_field.token_x_forcefield, sizeof(h_d_tmp.force_field.token_x_forcefield), "VXA.Simulator.ForceField.x_forcefield", pt_merged);
        ParseMathTree(h_d_tmp.force_field.token_y_forcefield, sizeof(h_d_tmp.force_field.token_y_forcefield), "VXA.Simulator.ForceField.y_forcefield", pt_merged);
        ParseMathTree(h_d_tmp.force_field.token_z_forcefield, sizeof(h_d_tmp.force_field.token_z_forcefield), "VXA.Simulator.ForceField.z_forcefield", pt_merged);

        // h_d_tmp.EnableTargetCloseness = pt_merged.get<int>("VXA.Simulator.EnableTargetCloseness", 0); abandoned.
        h_d_tmp.SavePositionOfAllVoxels = pt_merged.get<int>("VXA.Simulator.SavePositionOfAllVoxels", 0);
        h_d_tmp.MaxDistInVoxelLengthsToCountAsPair = pt_merged.get<double>("VXA.Simulator.MaxDistInVoxelLengthsToCountAsPair", 0);

        h_d_tmp.EnableCilia = pt_merged.get<int>("VXA.Simulator.EnableCilia", 0);
        h_d_tmp.RandomizeCiliaEvery = pt_merged.get<double>("VXA.Simulator.RandomizeCiliaEvery", 0);  // sam
        h_d_tmp.RandomSeed = pt_merged.get<double>("VXA.Simulator.RandomSeed", 0);  // sam
        h_d_tmp.EnableSignals = pt_merged.get<int>("VXA.Simulator.EnableSignals", 0);
        h_d_tmp.ReplenishDebrisEvery = pt_merged.get<double>("VXA.Simulator.ReplenishDebrisEvery", 0);  // sam
        h_d_tmp.ComputeLargestSitckyGroupForFirstRound = pt_merged.get<bool>("VXA.Simulator.ComputeLargestSitckyGroupForFirstRound", false);  // sam

        h_d_tmp.VerboseMode = pt_merged.get<bool>("VXA.Simulator.ThoroughTest.VerboseMode", false); // sam: off by default
        h_d_tmp.SkipThoroughTest = pt_merged.get<bool>("VXA.Simulator.ThoroughTest.SkipTest", true);
        h_d_tmp.ThoroughTestStepSize = pt_merged.get<unsigned int>("VXA.Simulator.ThoroughTest.TestStepSize", 100);
        h_d_tmp.ThoroughTestStartAt = pt_merged.get<unsigned int>("VXA.Simulator.ThoroughTest.TestStartAt", 0);

        // for Secondary Experiment
        h_d_tmp.SecondaryExperiment = pt_merged.get<int>("VXA.Simulator.SecondaryExperiment", 0);
        h_d_tmp.SelfReplication = pt_merged.get<int>("VXA.Simulator.SelfReplication", 0);  // sam
        h_d_tmp.WorldSize = pt_merged.get<int>("VXA.Simulator.WorldSize", 1);  // sam
        h_d_tmp.WallForce = pt_merged.get<double>("VXA.Simulator.WallForce", 0);  // sam
        h_d_tmp.BotMat = pt_merged.get<int>("VXA.Simulator.BotMat", 1);  // sam
        h_d_tmp.RemoveMat = pt_merged.get<int>("VXA.Simulator.RemoveMat", 1);  // sam
        h_d_tmp.DebrisMat = pt_merged.get<int>("VXA.Simulator.DebrisMat", 2);  // sam
        h_d_tmp.SpaceBetweenDebris = pt_merged.get<int>("VXA.Simulator.SpaceBetweenDebris", 2);  // sam
        h_d_tmp.DebrisHeight = pt_merged.get<int>("VXA.Simulator.DebrisHeight", 1);  // sam
        h_d_tmp.DebrisConcentration = pt_merged.get<int>("VXA.Simulator.DebrisConcentration", 1);  // sam
        h_d_tmp.DetachStringyBodiesEvery = pt_merged.get<double>("VXA.Simulator.DetachStringyBodiesEvery", 0);  // sam
        h_d_tmp.nonStickyTimeAfterStringyBodyDetach = pt_merged.get<double>("VXA.Simulator.nonStickyTimeAfterStringyBodyDetach", 0);  // sam
        h_d_tmp.DetachProbability = pt_merged.get<double>("VXA.Simulator.DetachProbability", 0);  // sam
        
        h_d_tmp.ReinitializeInitialPositionAfterThisManySeconds = pt_merged.get<double>("VXA.Simulator.ReinitializeInitialPositionAfterThisManySeconds", 0.0);
        h_d_tmp.SettleTimeBeforeNextRoundOfReplication = pt_merged.get<double>("VXA.Simulator.SettleTimeBeforeNextRoundOfReplication", 0.0); // sam
        h_d_tmp.MinimumBotSize = pt_merged.get<int>("VXA.Simulator.MinimumBotSize", 0); // sam

        h_d_tmp.CiliaFracAfterFirstRound = pt_merged.get<int>("VXA.Simulator.CiliaFracAfterFirstRound", 1); 

        h_d_tmp.MaxNewVoxelsAddedMidSim = pt_merged.get<int>("VXA.Simulator.MaxNewVoxelsAddedMidSim", 0); // sam

        h_d_tmp.EnableExpansion = pt_merged.get<int>("VXA.Simulator.EnableExpansion", 0);

        h_d_tmp.EnableSurfaceGrowth = pt_merged.get<int>("VXA.Simulator.SurfaceGrowth.EnableGrowth", 0);
        h_d_tmp.SurfaceGrowth_Interval = pt_merged.get<double>("VXA.Simulator.SurfaceGrowth.GrowInterval", 1);
        h_d_tmp.SurfaceGrowth_Rate = pt_merged.get<double>("VXA.Simulator.SurfaceGrowth.GrowRate", 0.1);

        HeapSize = pt_merged.get<double>("VXA.GPU.HeapSize", 0.5);
        if (HeapSize > 1.0) {
            HeapSize = 0.99;
        }
        if (HeapSize < 0.01) {
            HeapSize = 0.01;
        }

        VcudaMemcpy(d_voxelyze_3s[device_index] + i, &h_d_tmp, sizeof(VX3_VoxelyzeKernel), hipMemcpyHostToDevice);
        i++;
    }
}

// GPU Heap is for in-kernel malloc(). Refer to
// https://stackoverflow.com/a/34795830/7001199
void VX3_SimulationManager::enlargeGPUHeapSize() {
    size_t HeapSizeInBytes;
    size_t free, total;

    // Query default limitations:
    // size_t pValue;
    // hipDeviceGetLimit(&pValue, hipLimitMallocHeapSize);
    // printf("hipDeviceGetLimit, hipLimitMallocHeapSize, %ld.\n", pValue);
    // hipDeviceGetLimit(&pValue, hipLimitStackSize);
    // printf("hipDeviceGetLimit, hipLimitStackSize, %ld.\n", pValue);
    // hipDeviceGetLimit(&pValue, hipLimitPrintfFifoSize);
    // printf("hipDeviceGetLimit, hipLimitPrintfFifoSize, %ld.\n", pValue);

    VcudaMemGetInfo(&free, &total);
    printf("Total GPU memory %ld bytes.\n", total);
    HeapSizeInBytes = HeapSize * total; // add some additional size
    printf("Set GPU heap size to be %ld bytes.\n", HeapSizeInBytes);
    VcudaDeviceSetLimit(hipLimitMallocHeapSize,
                        HeapSizeInBytes); // Set Heap Memory to 1G, instead of merely 8M.

    // if "Lane User Stack Overflow" ocurs, maybe Stack Size too small, can try this:
    // VcudaDeviceSetLimit(hipLimitStackSize, 1024*2);
    // VcudaDeviceSetLimit(hipLimitPrintfFifoSize, 2048);
}

void VX3_SimulationManager::startKernel(int num_simulation, int device_index) {
    int threadsPerBlock = 512;
    int numBlocks = (num_simulation + threadsPerBlock - 1) / threadsPerBlock;
    if (numBlocks == 1)
        threadsPerBlock = num_simulation;
    // printf("Starting kernel on device %d. passing d_voxelyze_3s[device_index]
    // %p.\n", device_index, d_voxelyze_3s[device_index]);
    // VX3_VoxelyzeKernel *result_voxelyze_kernel = (VX3_VoxelyzeKernel
    // *)malloc(
    //     num_simulation * sizeof(VX3_VoxelyzeKernel));
    // VcudaMemcpy(result_voxelyze_kernel, d_voxelyze_3s[device_index],
    //             num_simulation * sizeof(VX3_VoxelyzeKernel),
    //             hipMemcpyDeviceToHost);
    enlargeGPUHeapSize();
    printf("Start %d Simulations.\n", num_simulation);
    // CUDA_Simulation<<<numBlocks, threadsPerBlock>>>(d_voxelyze_3s[device_index], num_simulation, device_index);
    sequential_CUDA_Simulation<<<1, 1>>>(d_voxelyze_3s[device_index], num_simulation, device_index);
    CUDA_CHECK_AFTER_CALL();
    // VcudaDeviceSynchronize();
    // NO!! We don't need to synchronize here! It will be super slow to process large number of simulations!
}

void VX3_SimulationManager::collectResults(int num_simulation, int device_index) {
    // insert results to h_results
    VX3_VoxelyzeKernel *result_voxelyze_kernel = (VX3_VoxelyzeKernel *)malloc(num_simulation * sizeof(VX3_VoxelyzeKernel));
    VcudaMemcpy(result_voxelyze_kernel, d_voxelyze_3s[device_index], num_simulation * sizeof(VX3_VoxelyzeKernel), hipMemcpyDeviceToHost);
    for (int i = 0; i < num_simulation; i++) {
        VX3_SimulationResult tmp;
        tmp.currentTime = result_voxelyze_kernel[i].currentTime;
        tmp.fitness_score = result_voxelyze_kernel[i].fitness_score;
        // tmp.x = result_voxelyze_kernel[i].currentCenterOfMass.x;
        // tmp.y = result_voxelyze_kernel[i].currentCenterOfMass.y;
        // tmp.z = result_voxelyze_kernel[i].currentCenterOfMass.z;
        // result_voxelyze_kernel[i].initialCenterOfMass.copyTo(tmp.initialCenterOfMass);
        // result_voxelyze_kernel[i].currentCenterOfMass.copyTo(tmp.currentCenterOfMass);

        tmp.largestStickyGroupSize = result_voxelyze_kernel[i].largestStickyGroupSize;
        // tmp.numRealLinks = result_voxelyze_kernel[i].numRealLinks;
        // tmp.numClosePairs = result_voxelyze_kernel[i].numClosePairs;
        // tmp.voxSize = result_voxelyze_kernel[i].voxSize;
        // tmp.num_voxel = result_voxelyze_kernel[i].num_d_voxels;
        tmp.vxa_filename = result_voxelyze_kernel[i].vxa_filename;
        
        if (true) {
            tmp.numLightsOn = result_voxelyze_kernel[i].numLightsOn;
            tmp.pileSize01 = result_voxelyze_kernel[i].pileSize01;
            tmp.pileSize02 = result_voxelyze_kernel[i].pileSize02;
            tmp.pileSize03 = result_voxelyze_kernel[i].pileSize03;
            tmp.pileSize04 = result_voxelyze_kernel[i].pileSize04;
            tmp.pileSize05 = result_voxelyze_kernel[i].pileSize05;
            tmp.pileSize06 = result_voxelyze_kernel[i].pileSize06;
            tmp.pileSize07 = result_voxelyze_kernel[i].pileSize07;
            tmp.pileSize08 = result_voxelyze_kernel[i].pileSize08;
            tmp.pileSize09 = result_voxelyze_kernel[i].pileSize09;
            tmp.pileSize10 = result_voxelyze_kernel[i].pileSize10;
            tmp.pileSize11 = result_voxelyze_kernel[i].pileSize11;
            tmp.pileSize12 = result_voxelyze_kernel[i].pileSize12;
            tmp.pileSize13 = result_voxelyze_kernel[i].pileSize13;
            tmp.pileSize14 = result_voxelyze_kernel[i].pileSize14;
            tmp.pileSize15 = result_voxelyze_kernel[i].pileSize15;
            tmp.pileSize16 = result_voxelyze_kernel[i].pileSize16;
            tmp.pileSize17 = result_voxelyze_kernel[i].pileSize17;
            tmp.pileSize18 = result_voxelyze_kernel[i].pileSize18;
        }
        
        // VX3_Voxel *tmp_v;
        // tmp_v = (VX3_Voxel *)malloc(result_voxelyze_kernel[i].num_d_voxels * sizeof(VX3_Voxel));
        // VcudaMemcpy(tmp_v, result_voxelyze_kernel[i].d_voxels, result_voxelyze_kernel[i].num_d_voxels * sizeof(VX3_Voxel), hipMemcpyDeviceToHost);
        // tmp.SavePositionOfAllVoxels = result_voxelyze_kernel[i].SavePositionOfAllVoxels;
        // VX3_Vec3D<> *tmp_init;
        // tmp_init = (VX3_Vec3D<> *)malloc(result_voxelyze_kernel[i].num_d_init_voxels * sizeof(VX3_Vec3D<>));
        // VcudaMemcpy(tmp_init, result_voxelyze_kernel[i].d_initialPosition, result_voxelyze_kernel[i].num_d_init_voxels * sizeof(VX3_Vec3D<>), hipMemcpyDeviceToHost);
        // tmp.num_measured_voxel = 0;
        // tmp.total_distance_of_all_voxels = 0.0;
        // for (int j = 0; j < result_voxelyze_kernel[i].num_d_init_voxels; j++) {
        //     tmp.voxel_init_pos.push_back(Vec3D<>(tmp_init[j].x, tmp_init[j].y, tmp_init[j].z));
        //     tmp.voxel_position.push_back(Vec3D<>(tmp_v[j].pos.x, tmp_v[j].pos.y, tmp_v[j].pos.z));
        //     tmp.voxel_mats.push_back(tmp_v[j].matid);
        //     if (tmp_v[j].isMeasured) {
        //         tmp.num_measured_voxel++;
        //         tmp.total_distance_of_all_voxels += tmp.voxel_position.back().Dist(tmp.voxel_init_pos.back());
        //     }
        // }
        // delete tmp_v;

        // tmp.computeFitness();
        h_results.push_back(tmp);
    }
}

void VX3_SimulationManager::sortResults() { sort(h_results.begin(), h_results.end(), VX3_SimulationResult::compareFitnessScore); }
